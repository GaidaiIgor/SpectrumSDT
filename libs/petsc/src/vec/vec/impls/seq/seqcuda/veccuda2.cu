#include "hip/hip_runtime.h"
/*
   Implements the sequential cuda vectors.
*/

#define PETSC_SKIP_SPINLOCK
#define PETSC_SKIP_CXX_COMPLEX_FIX

#include <petscconf.h>
#include <petsc/private/vecimpl.h>
#include <../src/vec/vec/impls/dvecimpl.h>
#include <../src/vec/vec/impls/seq/seqcuda/cudavecimpl.h>

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

/*
    Allocates space for the vector array on the GPU if it does not exist.
    Does NOT change the PetscCUDAFlag for the vector
    Does NOT zero the CUDA array

 */
PetscErrorCode VecCUDAAllocateCheck(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;
  Vec_CUDA       *veccuda;
  PetscBool      option_set;

  PetscFunctionBegin;
  if (!v->spptr) {
    PetscReal pinned_memory_min;
    ierr = PetscMalloc(sizeof(Vec_CUDA),&v->spptr);CHKERRQ(ierr);
    veccuda = (Vec_CUDA*)v->spptr;
    err = hipMalloc((void**)&veccuda->GPUarray_allocated,sizeof(PetscScalar)*((PetscBLASInt)v->map->n));CHKERRCUDA(err);
    veccuda->GPUarray = veccuda->GPUarray_allocated;
    veccuda->stream = 0;  /* using default stream */
    if (v->offloadmask == PETSC_OFFLOAD_UNALLOCATED) {
      if (v->data && ((Vec_Seq*)v->data)->array) {
        v->offloadmask = PETSC_OFFLOAD_CPU;
      } else {
        v->offloadmask = PETSC_OFFLOAD_GPU;
      }
    }
    pinned_memory_min = 0;

    /* Need to parse command line for minimum size to use for pinned memory allocations on host here.
       Note: This same code duplicated in VecCreate_SeqCUDA_Private() and VecCreate_MPICUDA_Private(). Is there a good way to avoid this? */
    ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)v),((PetscObject)v)->prefix,"VECCUDA Options","Vec");CHKERRQ(ierr);
    ierr = PetscOptionsReal("-vec_pinned_memory_min","Minimum size (in bytes) for an allocation to use pinned memory on host","VecSetPinnedMemoryMin",pinned_memory_min,&pinned_memory_min,&option_set);CHKERRQ(ierr);
    if (option_set) v->minimum_bytes_pinned_memory = pinned_memory_min;
    ierr = PetscOptionsEnd();CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/* Copies a vector from the CPU to the GPU unless we already have an up-to-date copy on the GPU */
PetscErrorCode VecCUDACopyToGPU(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;
  Vec_CUDA       *veccuda;
  PetscScalar    *varray;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDAAllocateCheck(v);CHKERRQ(ierr);
  if (v->offloadmask == PETSC_OFFLOAD_CPU) {
    ierr               = PetscLogEventBegin(VEC_CUDACopyToGPU,v,0,0,0);CHKERRQ(ierr);
    veccuda            = (Vec_CUDA*)v->spptr;
    varray             = veccuda->GPUarray;
    err                = hipMemcpy(varray,((Vec_Seq*)v->data)->array,v->map->n*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
    ierr               = PetscLogCpuToGpu((v->map->n)*sizeof(PetscScalar));CHKERRQ(ierr);
    ierr               = PetscLogEventEnd(VEC_CUDACopyToGPU,v,0,0,0);CHKERRQ(ierr);
    v->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecCUDACopyToGPUSome(Vec v, PetscCUDAIndices ci,ScatterMode mode)
{
  PetscScalar    *varray;
  PetscErrorCode ierr;
  hipError_t    err;
  PetscScalar    *cpuPtr, *gpuPtr;
  Vec_Seq        *s;
  VecScatterCUDAIndices_PtoP ptop_scatter = (VecScatterCUDAIndices_PtoP)ci->scatter;
  PetscInt       lowestIndex,n;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDAAllocateCheck(v);CHKERRQ(ierr);
  if (v->offloadmask == PETSC_OFFLOAD_CPU) {
    s = (Vec_Seq*)v->data;
    if (mode & SCATTER_REVERSE) {
      lowestIndex = ptop_scatter->sendLowestIndex;
      n           = ptop_scatter->ns;
    } else {
      lowestIndex = ptop_scatter->recvLowestIndex;
      n           = ptop_scatter->nr;
    }

    ierr   = PetscLogEventBegin(VEC_CUDACopyToGPUSome,v,0,0,0);CHKERRQ(ierr);
    varray = ((Vec_CUDA*)v->spptr)->GPUarray;
    gpuPtr = varray + lowestIndex;
    cpuPtr = s->array + lowestIndex;

    /* Note : this code copies the smallest contiguous chunk of data
       containing ALL of the indices */
    err = hipMemcpy(gpuPtr,cpuPtr,n*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
    ierr = PetscLogCpuToGpu(n*sizeof(PetscScalar));CHKERRQ(ierr);

    /* Set the buffer states */
    v->offloadmask = PETSC_OFFLOAD_BOTH;
    ierr = PetscLogEventEnd(VEC_CUDACopyToGPUSome,v,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}


/*
     VecCUDACopyFromGPU - Copies a vector from the GPU to the CPU unless we already have an up-to-date copy on the CPU
*/
PetscErrorCode VecCUDACopyFromGPU(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;
  Vec_CUDA       *veccuda;
  PetscScalar    *varray;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDAAllocateCheckHost(v);CHKERRQ(ierr);
  if (v->offloadmask == PETSC_OFFLOAD_GPU) {
    ierr               = PetscLogEventBegin(VEC_CUDACopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    veccuda            = (Vec_CUDA*)v->spptr;
    varray             = veccuda->GPUarray;
    err                = hipMemcpy(((Vec_Seq*)v->data)->array,varray,v->map->n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(err);
    ierr               = PetscLogGpuToCpu((v->map->n)*sizeof(PetscScalar));CHKERRQ(ierr);
    ierr               = PetscLogEventEnd(VEC_CUDACopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    v->offloadmask     = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

/* Note that this function only copies *some* of the values up from the GPU to CPU,
   which means that we need recombine the data at some point before using any of the standard functions.
   We could add another few flag-types to keep track of this, or treat things like VecGetArray VecRestoreArray
   where you have to always call in pairs
*/
PetscErrorCode VecCUDACopyFromGPUSome(Vec v, PetscCUDAIndices ci,ScatterMode mode)
{
  const PetscScalar *varray, *gpuPtr;
  PetscErrorCode    ierr;
  hipError_t       err;
  PetscScalar       *cpuPtr;
  Vec_Seq           *s;
  VecScatterCUDAIndices_PtoP ptop_scatter = (VecScatterCUDAIndices_PtoP)ci->scatter;
  PetscInt          lowestIndex,n;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDAAllocateCheckHost(v);CHKERRQ(ierr);
  if (v->offloadmask == PETSC_OFFLOAD_GPU) {
    ierr   = PetscLogEventBegin(VEC_CUDACopyFromGPUSome,v,0,0,0);CHKERRQ(ierr);
    if (mode & SCATTER_REVERSE) {
      lowestIndex = ptop_scatter->recvLowestIndex;
      n           = ptop_scatter->nr;
    } else {
      lowestIndex = ptop_scatter->sendLowestIndex;
      n           = ptop_scatter->ns;
    }

    varray=((Vec_CUDA*)v->spptr)->GPUarray;
    s = (Vec_Seq*)v->data;
    gpuPtr = varray + lowestIndex;
    cpuPtr = s->array + lowestIndex;

    /* Note : this code copies the smallest contiguous chunk of data
       containing ALL of the indices */
    err = hipMemcpy(cpuPtr,gpuPtr,n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(err);
    ierr = PetscLogGpuToCpu(n*sizeof(PetscScalar));CHKERRQ(ierr);

    ierr = VecCUDARestoreArrayRead(v,&varray);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(VEC_CUDACopyFromGPUSome,v,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*MC
   VECSEQCUDA - VECSEQCUDA = "seqcuda" - The basic sequential vector, modified to use CUDA

   Options Database Keys:
. -vec_type seqcuda - sets the vector type to VECSEQCUDA during a call to VecSetFromOptions()

  Level: beginner

.seealso: VecCreate(), VecSetType(), VecSetFromOptions(), VecCreateSeqWithArray(), VECMPI, VecType, VecCreateMPI(), VecCreateSeq(), VecSetPinnedMemoryMin()
M*/

PetscErrorCode VecAYPX_SeqCUDA(Vec yin,PetscScalar alpha,Vec xin)
{
  const PetscScalar *xarray;
  PetscScalar       *yarray;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  PetscScalar       sone=1.0;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;
  hipError_t       err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(yin->map->n,&bn);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArray(yin,&yarray);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (alpha == (PetscScalar)0.0) {
    err = hipMemcpy(yarray,xarray,bn*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
  } else if (alpha == (PetscScalar)1.0) {
    cberr = cublasXaxpy(cublasv2handle,bn,&alpha,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    ierr = PetscLogGpuFlops(1.0*yin->map->n);CHKERRQ(ierr);
  } else {
    cberr = cublasXscal(cublasv2handle,bn,&alpha,yarray,one);CHKERRCUBLAS(cberr);
    cberr = cublasXaxpy(cublasv2handle,bn,&sone,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    ierr = PetscLogGpuFlops(2.0*yin->map->n);CHKERRQ(ierr);
  }
  err  = WaitForGPU();CHKERRCUDA(err);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArray(yin,&yarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecAXPY_SeqCUDA(Vec yin,PetscScalar alpha,Vec xin)
{
  const PetscScalar *xarray;
  PetscScalar       *yarray;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;
  hipError_t       err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  if (alpha != (PetscScalar)0.0) {
    ierr = PetscBLASIntCast(yin->map->n,&bn);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArray(yin,&yarray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXaxpy(cublasv2handle,bn,&alpha,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    err  = WaitForGPU();CHKERRCUDA(err);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArray(yin,&yarray);CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(2.0*yin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecPointwiseDivide_SeqCUDA(Vec win, Vec xin, Vec yin)
{
  PetscInt                              n = xin->map->n;
  const PetscScalar                     *xarray=NULL,*yarray=NULL;
  PetscScalar                           *warray=NULL;
  thrust::device_ptr<const PetscScalar> xptr,yptr;
  thrust::device_ptr<PetscScalar>       wptr;
  PetscErrorCode                        ierr;
  hipError_t                           err;

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayWrite(win,&warray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  try {
    wptr = thrust::device_pointer_cast(warray);
    xptr = thrust::device_pointer_cast(xarray);
    yptr = thrust::device_pointer_cast(yarray);
    thrust::transform(xptr,xptr+n,yptr,wptr,thrust::divides<PetscScalar>());
    err  = WaitForGPU();CHKERRCUDA(err);
  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Thrust error: %s", ex);
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(n);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(win,&warray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecWAXPY_SeqCUDA(Vec win,PetscScalar alpha,Vec xin, Vec yin)
{
  const PetscScalar *xarray=NULL,*yarray=NULL;
  PetscScalar       *warray=NULL;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;
  hipError_t       err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(win->map->n,&bn);CHKERRQ(ierr);
  if (alpha == (PetscScalar)0.0) {
    ierr = VecCopy_SeqCUDA(yin,win);CHKERRQ(ierr);
  } else {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayWrite(win,&warray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    err = hipMemcpy(warray,yarray,win->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
    cberr = cublasXaxpy(cublasv2handle,bn,&alpha,xarray,one,warray,one);CHKERRCUBLAS(cberr);
    err  = WaitForGPU();CHKERRCUDA(err);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(2*win->map->n);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayWrite(win,&warray);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecMAXPY_SeqCUDA(Vec xin, PetscInt nv,const PetscScalar *alpha,Vec *y)
{
  PetscErrorCode ierr;
  hipError_t    err;
  PetscInt       n = xin->map->n,j,j_rem;
  PetscScalar    alpha0,alpha1,alpha2,alpha3;

  PetscFunctionBegin;
  ierr = PetscLogGpuFlops(nv*2.0*n);CHKERRQ(ierr);
  switch (j_rem=nv&0x3) {
    case 3:
      alpha0 = alpha[0];
      alpha1 = alpha[1];
      alpha2 = alpha[2];
      alpha += 3;
      ierr   = VecAXPY_SeqCUDA(xin,alpha0,y[0]);CHKERRQ(ierr);
      ierr   = VecAXPY_SeqCUDA(xin,alpha1,y[1]);CHKERRQ(ierr);
      ierr   = VecAXPY_SeqCUDA(xin,alpha2,y[2]);CHKERRQ(ierr);
      y   += 3;
      break;
    case 2:
      alpha0 = alpha[0];
      alpha1 = alpha[1];
      alpha +=2;
      ierr   = VecAXPY_SeqCUDA(xin,alpha0,y[0]);CHKERRQ(ierr);
      ierr   = VecAXPY_SeqCUDA(xin,alpha1,y[1]);CHKERRQ(ierr);
      y +=2;
      break;
    case 1:
      alpha0 = *alpha++;
      ierr   = VecAXPY_SeqCUDA(xin,alpha0,y[0]);CHKERRQ(ierr);
      y     +=1;
      break;
  }
  for (j=j_rem; j<nv; j+=4) {
    alpha0 = alpha[0];
    alpha1 = alpha[1];
    alpha2 = alpha[2];
    alpha3 = alpha[3];
    alpha += 4;
    ierr   = VecAXPY_SeqCUDA(xin,alpha0,y[0]);CHKERRQ(ierr);
    ierr   = VecAXPY_SeqCUDA(xin,alpha1,y[1]);CHKERRQ(ierr);
    ierr   = VecAXPY_SeqCUDA(xin,alpha2,y[2]);CHKERRQ(ierr);
    ierr   = VecAXPY_SeqCUDA(xin,alpha3,y[3]);CHKERRQ(ierr);
    y   += 4;
  }
  err  = WaitForGPU();CHKERRCUDA(err);
  PetscFunctionReturn(0);
}

PetscErrorCode VecDot_SeqCUDA(Vec xin,Vec yin,PetscScalar *z)
{
  const PetscScalar *xarray,*yarray;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cerr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(yin->map->n,&bn);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  /* arguments y, x are reversed because BLAS complex conjugates the first argument, PETSc the second */
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  cerr = cublasXdot(cublasv2handle,bn,yarray,one,xarray,one,z);CHKERRCUBLAS(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  if (xin->map->n >0) {
    ierr = PetscLogGpuFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

//
// CUDA kernels for MDot to follow
//

// set work group size to be a power of 2 (128 is usually a good compromise between portability and speed)
#define MDOT_WORKGROUP_SIZE 128
#define MDOT_WORKGROUP_NUM  128

#if !defined(PETSC_USE_COMPLEX)
// M = 2:
__global__ void VecMDot_SeqCUDA_kernel2(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,
                                        PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[2*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
  }
  tmp_buffer[threadIdx.x]                       = group_sum0;
  tmp_buffer[threadIdx.x + MDOT_WORKGROUP_SIZE] = group_sum1;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                      ] += tmp_buffer[threadIdx.x+stride                      ];
      tmp_buffer[threadIdx.x + MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x]             = tmp_buffer[0];
    group_results[blockIdx.x + gridDim.x] = tmp_buffer[MDOT_WORKGROUP_SIZE];
  }
}

// M = 3:
__global__ void VecMDot_SeqCUDA_kernel3(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,const PetscScalar *y2,
                                        PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[3*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  PetscScalar group_sum2 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
    group_sum2 += entry_x * y2[i];
  }
  tmp_buffer[threadIdx.x]                           = group_sum0;
  tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] = group_sum1;
  tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] = group_sum2;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                          ] += tmp_buffer[threadIdx.x+stride                          ];
      tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride +     MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 2 * MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x                ] = tmp_buffer[0];
    group_results[blockIdx.x +     gridDim.x] = tmp_buffer[    MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 2 * gridDim.x] = tmp_buffer[2 * MDOT_WORKGROUP_SIZE];
  }
}

// M = 4:
__global__ void VecMDot_SeqCUDA_kernel4(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,const PetscScalar *y2,const PetscScalar *y3,
                                        PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[4*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  PetscScalar group_sum2 = 0;
  PetscScalar group_sum3 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
    group_sum2 += entry_x * y2[i];
    group_sum3 += entry_x * y3[i];
  }
  tmp_buffer[threadIdx.x]                           = group_sum0;
  tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] = group_sum1;
  tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] = group_sum2;
  tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] = group_sum3;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                          ] += tmp_buffer[threadIdx.x+stride                          ];
      tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride +     MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 2 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 3 * MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x                ] = tmp_buffer[0];
    group_results[blockIdx.x +     gridDim.x] = tmp_buffer[    MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 2 * gridDim.x] = tmp_buffer[2 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 3 * gridDim.x] = tmp_buffer[3 * MDOT_WORKGROUP_SIZE];
  }
}

// M = 8:
__global__ void VecMDot_SeqCUDA_kernel8(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,const PetscScalar *y2,const PetscScalar *y3,
                                          const PetscScalar *y4,const PetscScalar *y5,const PetscScalar *y6,const PetscScalar *y7,
                                          PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[8*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  PetscScalar group_sum2 = 0;
  PetscScalar group_sum3 = 0;
  PetscScalar group_sum4 = 0;
  PetscScalar group_sum5 = 0;
  PetscScalar group_sum6 = 0;
  PetscScalar group_sum7 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
    group_sum2 += entry_x * y2[i];
    group_sum3 += entry_x * y3[i];
    group_sum4 += entry_x * y4[i];
    group_sum5 += entry_x * y5[i];
    group_sum6 += entry_x * y6[i];
    group_sum7 += entry_x * y7[i];
  }
  tmp_buffer[threadIdx.x]                           = group_sum0;
  tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] = group_sum1;
  tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] = group_sum2;
  tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] = group_sum3;
  tmp_buffer[threadIdx.x + 4 * MDOT_WORKGROUP_SIZE] = group_sum4;
  tmp_buffer[threadIdx.x + 5 * MDOT_WORKGROUP_SIZE] = group_sum5;
  tmp_buffer[threadIdx.x + 6 * MDOT_WORKGROUP_SIZE] = group_sum6;
  tmp_buffer[threadIdx.x + 7 * MDOT_WORKGROUP_SIZE] = group_sum7;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                          ] += tmp_buffer[threadIdx.x+stride                          ];
      tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride +     MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 2 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 3 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 4 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 4 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 5 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 5 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 6 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 6 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 7 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 7 * MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x                ] = tmp_buffer[0];
    group_results[blockIdx.x +     gridDim.x] = tmp_buffer[    MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 2 * gridDim.x] = tmp_buffer[2 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 3 * gridDim.x] = tmp_buffer[3 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 4 * gridDim.x] = tmp_buffer[4 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 5 * gridDim.x] = tmp_buffer[5 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 6 * gridDim.x] = tmp_buffer[6 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 7 * gridDim.x] = tmp_buffer[7 * MDOT_WORKGROUP_SIZE];
  }
}
#endif /* !defined(PETSC_USE_COMPLEX) */

PetscErrorCode VecMDot_SeqCUDA(Vec xin,PetscInt nv,const Vec yin[],PetscScalar *z)
{
  PetscErrorCode    ierr;
  PetscInt          i,n = xin->map->n,current_y_index = 0;
  const PetscScalar *xptr,*y0ptr,*y1ptr,*y2ptr,*y3ptr,*y4ptr,*y5ptr,*y6ptr,*y7ptr;
  PetscScalar       *group_results_gpu;
#if !defined(PETSC_USE_COMPLEX)
  PetscInt          j;
  PetscScalar       group_results_cpu[MDOT_WORKGROUP_NUM * 8]; // we process at most eight vectors in one kernel
#endif
  hipError_t    cuda_ierr;
  PetscBLASInt   one=1,bn;
  hipblasHandle_t cublasv2handle;
  hipblasStatus_t cberr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(xin->map->n,&bn);CHKERRQ(ierr);
  if (nv <= 0) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"Number of vectors provided to VecMDot_SeqCUDA not positive.");
  /* Handle the case of local size zero first */
  if (!xin->map->n) {
    for (i=0; i<nv; ++i) z[i] = 0;
    PetscFunctionReturn(0);
  }

  // allocate scratchpad memory for the results of individual work groups:
  cuda_ierr = hipMalloc((void**)&group_results_gpu, sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 8);CHKERRCUDA(cuda_ierr);

  ierr = VecCUDAGetArrayRead(xin,&xptr);CHKERRQ(ierr);

  while (current_y_index < nv)
  {
    switch (nv - current_y_index) {

      case 7:
      case 6:
      case 5:
      case 4:
        ierr = VecCUDAGetArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+3],&y3ptr);CHKERRQ(ierr);
        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y1ptr,one,xptr,one,&z[current_y_index+1]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y2ptr,one,xptr,one,&z[current_y_index+2]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y3ptr,one,xptr,one,&z[current_y_index+3]);CHKERRCUBLAS(cberr);
#else
        // run kernel:
        VecMDot_SeqCUDA_kernel4<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,y2ptr,y3ptr,n,group_results_gpu);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

        // copy results back to
        cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 4,hipMemcpyDeviceToHost);CHKERRCUDA(cuda_ierr);

        // sum group results into z:
        for (j=0; j<4; ++j) {
          z[current_y_index + j] = 0;
          for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
        }
#endif
        ierr = VecCUDARestoreArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+3],&y3ptr);CHKERRQ(ierr);
        current_y_index += 4;
        break;

      case 3:
        ierr = VecCUDAGetArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);

        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y1ptr,one,xptr,one,&z[current_y_index+1]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y2ptr,one,xptr,one,&z[current_y_index+2]);CHKERRCUBLAS(cberr);
#else
        // run kernel:
        VecMDot_SeqCUDA_kernel3<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,y2ptr,n,group_results_gpu);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

        // copy results back to
        cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 3,hipMemcpyDeviceToHost);CHKERRCUDA(cuda_ierr);

        // sum group results into z:
        for (j=0; j<3; ++j) {
          z[current_y_index + j] = 0;
          for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
        }
#endif
        ierr = VecCUDARestoreArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        current_y_index += 3;
        break;

      case 2:
        ierr = VecCUDAGetArrayRead(yin[current_y_index],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y1ptr,one,xptr,one,&z[current_y_index+1]);CHKERRCUBLAS(cberr);
#else
        // run kernel:
        VecMDot_SeqCUDA_kernel2<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,n,group_results_gpu);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

        // copy results back to
        cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 2,hipMemcpyDeviceToHost);CHKERRCUDA(cuda_ierr);

        // sum group results into z:
        for (j=0; j<2; ++j) {
          z[current_y_index + j] = 0;
          for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
        }
#endif
        ierr = VecCUDARestoreArrayRead(yin[current_y_index],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        current_y_index += 2;
        break;

      case 1:
        ierr = VecCUDAGetArrayRead(yin[current_y_index],&y0ptr);CHKERRQ(ierr);
        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index],&y0ptr);CHKERRQ(ierr);
        current_y_index += 1;
        break;

      default: // 8 or more vectors left
        ierr = VecCUDAGetArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+3],&y3ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+4],&y4ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+5],&y5ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+6],&y6ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+7],&y7ptr);CHKERRQ(ierr);
        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y1ptr,one,xptr,one,&z[current_y_index+1]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y2ptr,one,xptr,one,&z[current_y_index+2]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y3ptr,one,xptr,one,&z[current_y_index+3]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y4ptr,one,xptr,one,&z[current_y_index+4]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y5ptr,one,xptr,one,&z[current_y_index+5]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y6ptr,one,xptr,one,&z[current_y_index+6]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y7ptr,one,xptr,one,&z[current_y_index+7]);CHKERRCUBLAS(cberr);
#else
        // run kernel:
        VecMDot_SeqCUDA_kernel8<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,y2ptr,y3ptr,y4ptr,y5ptr,y6ptr,y7ptr,n,group_results_gpu);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

        // copy results back to
        cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 8,hipMemcpyDeviceToHost);CHKERRCUDA(cuda_ierr);

        // sum group results into z:
        for (j=0; j<8; ++j) {
          z[current_y_index + j] = 0;
          for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
        }
#endif
        ierr = VecCUDARestoreArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+3],&y3ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+4],&y4ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+5],&y5ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+6],&y6ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+7],&y7ptr);CHKERRQ(ierr);
        current_y_index += 8;
        break;
    }
  }
  ierr = VecCUDARestoreArrayRead(xin,&xptr);CHKERRQ(ierr);

  cuda_ierr = hipFree(group_results_gpu);CHKERRCUDA(cuda_ierr);
  ierr = PetscLogGpuFlops(PetscMax(nv*(2.0*n-1),0.0));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef MDOT_WORKGROUP_SIZE
#undef MDOT_WORKGROUP_NUM

PetscErrorCode VecSet_SeqCUDA(Vec xin,PetscScalar alpha)
{
  PetscInt                        n = xin->map->n;
  PetscScalar                     *xarray=NULL;
  thrust::device_ptr<PetscScalar> xptr;
  PetscErrorCode                  ierr;
  hipError_t                     err;

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayWrite(xin,&xarray);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (alpha == (PetscScalar)0.0) {
    err = hipMemset(xarray,0,n*sizeof(PetscScalar));CHKERRCUDA(err);
  } else {
    try {
      xptr = thrust::device_pointer_cast(xarray);
      thrust::fill(xptr,xptr+n,alpha);
    } catch (char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Thrust error: %s", ex);
    }
  }
  err  = WaitForGPU();CHKERRCUDA(err);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(xin,&xarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecScale_SeqCUDA(Vec xin,PetscScalar alpha)
{
  PetscScalar    *xarray;
  PetscErrorCode ierr;
  PetscBLASInt   one=1,bn;
  hipblasHandle_t cublasv2handle;
  hipblasStatus_t cberr;
  hipError_t    err;

  PetscFunctionBegin;
  if (alpha == (PetscScalar)0.0) {
    ierr = VecSet_SeqCUDA(xin,alpha);CHKERRQ(ierr);
    err  = WaitForGPU();CHKERRCUDA(err);
  } else if (alpha != (PetscScalar)1.0) {
    ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
    ierr = PetscBLASIntCast(xin->map->n,&bn);CHKERRQ(ierr);
    ierr = VecCUDAGetArray(xin,&xarray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXscal(cublasv2handle,bn,&alpha,xarray,one);CHKERRCUBLAS(cberr);
    ierr = VecCUDARestoreArray(xin,&xarray);CHKERRQ(ierr);
    err  = WaitForGPU();CHKERRCUDA(err);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  }
  ierr = PetscLogGpuFlops(xin->map->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecTDot_SeqCUDA(Vec xin,Vec yin,PetscScalar *z)
{
  const PetscScalar *xarray,*yarray;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cerr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(xin->map->n,&bn);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  cerr = cublasXdotu(cublasv2handle,bn,xarray,one,yarray,one,z);CHKERRCUBLAS(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  if (xin->map->n > 0) {
    ierr = PetscLogGpuFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecCopy_SeqCUDA(Vec xin,Vec yin)
{
  const PetscScalar *xarray;
  PetscScalar       *yarray;
  PetscErrorCode    ierr;
  hipError_t       err;

  PetscFunctionBegin;
  if (xin != yin) {
    if (xin->offloadmask == PETSC_OFFLOAD_GPU) {
      PetscBool yiscuda;

      ierr = PetscObjectTypeCompareAny((PetscObject)yin,&yiscuda,VECSEQCUDA,VECMPICUDA,"");CHKERRQ(ierr);
      ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
      if (yiscuda) {
        ierr = VecCUDAGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
      } else {
        ierr = VecGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
      }
      ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
      if (yiscuda) {
        err = hipMemcpy(yarray,xarray,yin->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
      } else {
        err = hipMemcpy(yarray,xarray,yin->map->n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(err);
      }
      err  = WaitForGPU();CHKERRCUDA(err);
      ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
      ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
      if (yiscuda) {
        ierr = VecCUDARestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);
      } else {
        ierr = VecRestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);
      }
    } else if (xin->offloadmask == PETSC_OFFLOAD_CPU) {
      /* copy in CPU if we are on the CPU */
      ierr = VecCopy_SeqCUDA_Private(xin,yin);CHKERRQ(ierr);
    } else if (xin->offloadmask == PETSC_OFFLOAD_BOTH) {
      /* if xin is valid in both places, see where yin is and copy there (because it's probably where we'll want to next use it) */
      if (yin->offloadmask == PETSC_OFFLOAD_CPU) {
        /* copy in CPU */
        ierr = VecCopy_SeqCUDA_Private(xin,yin);CHKERRQ(ierr);
      } else if (yin->offloadmask == PETSC_OFFLOAD_GPU) {
        /* copy in GPU */
        ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
        err  = hipMemcpy(yarray,xarray,yin->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);
      } else if (yin->offloadmask == PETSC_OFFLOAD_BOTH) {
        /* xin and yin are both valid in both places (or yin was unallocated before the earlier call to allocatecheck
           default to copy in GPU (this is an arbitrary choice) */
        ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
        err  = hipMemcpy(yarray,xarray,yin->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);
      } else {
        ierr = VecCopy_SeqCUDA_Private(xin,yin);CHKERRQ(ierr);
      }
    }
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecSwap_SeqCUDA(Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscBLASInt   one = 1,bn;
  PetscScalar    *xarray,*yarray;
  hipblasHandle_t cublasv2handle;
  hipblasStatus_t cberr;
  hipError_t    err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(xin->map->n,&bn);CHKERRQ(ierr);
  if (xin != yin) {
    ierr = VecCUDAGetArray(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArray(yin,&yarray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXswap(cublasv2handle,bn,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    err  = WaitForGPU();CHKERRCUDA(err);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = VecCUDARestoreArray(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArray(yin,&yarray);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecAXPBY_SeqCUDA(Vec yin,PetscScalar alpha,PetscScalar beta,Vec xin)
{
  PetscErrorCode    ierr;
  PetscScalar       a = alpha,b = beta;
  const PetscScalar *xarray;
  PetscScalar       *yarray;
  PetscBLASInt      one = 1, bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;
  hipError_t       err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(yin->map->n,&bn);CHKERRQ(ierr);
  if (a == (PetscScalar)0.0) {
    ierr = VecScale_SeqCUDA(yin,beta);CHKERRQ(ierr);
  } else if (b == (PetscScalar)1.0) {
    ierr = VecAXPY_SeqCUDA(yin,alpha,xin);CHKERRQ(ierr);
  } else if (a == (PetscScalar)1.0) {
    ierr = VecAYPX_SeqCUDA(yin,beta,xin);CHKERRQ(ierr);
  } else if (b == (PetscScalar)0.0) {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArray(yin,&yarray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    err = hipMemcpy(yarray,xarray,yin->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
    cberr = cublasXscal(cublasv2handle,bn,&alpha,yarray,one);CHKERRCUBLAS(cberr);
    err  = WaitForGPU();CHKERRCUDA(err);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(xin->map->n);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArray(yin,&yarray);CHKERRQ(ierr);
  } else {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArray(yin,&yarray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXscal(cublasv2handle,bn,&beta,yarray,one);CHKERRCUBLAS(cberr);
    cberr = cublasXaxpy(cublasv2handle,bn,&alpha,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArray(yin,&yarray);CHKERRQ(ierr);
    err  = WaitForGPU();CHKERRCUDA(err);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(3.0*xin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecAXPBYPCZ_SeqCUDA(Vec zin,PetscScalar alpha,PetscScalar beta,PetscScalar gamma,Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  hipError_t    err;
  PetscInt       n = zin->map->n;

  PetscFunctionBegin;
  if (gamma == (PetscScalar)1.0) {
    /* z = ax + b*y + z */
    ierr = VecAXPY_SeqCUDA(zin,alpha,xin);CHKERRQ(ierr);
    ierr = VecAXPY_SeqCUDA(zin,beta,yin);CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(4.0*n);CHKERRQ(ierr);
  } else {
    /* z = a*x + b*y + c*z */
    ierr = VecScale_SeqCUDA(zin,gamma);CHKERRQ(ierr);
    ierr = VecAXPY_SeqCUDA(zin,alpha,xin);CHKERRQ(ierr);
    ierr = VecAXPY_SeqCUDA(zin,beta,yin);CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(5.0*n);CHKERRQ(ierr);
  }
  err  = WaitForGPU();CHKERRCUDA(err);
  PetscFunctionReturn(0);
}

PetscErrorCode VecPointwiseMult_SeqCUDA(Vec win,Vec xin,Vec yin)
{
  PetscInt                              n = win->map->n;
  const PetscScalar                     *xarray,*yarray;
  PetscScalar                           *warray;
  thrust::device_ptr<const PetscScalar> xptr,yptr;
  thrust::device_ptr<PetscScalar>       wptr;
  PetscErrorCode                        ierr;
  hipError_t                           err;

  PetscFunctionBegin;
  ierr = VecCUDAGetArray(win,&warray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr); 
  try {
    wptr = thrust::device_pointer_cast(warray);
    xptr = thrust::device_pointer_cast(xarray);
    yptr = thrust::device_pointer_cast(yarray);
    thrust::transform(xptr,xptr+n,yptr,wptr,thrust::multiplies<PetscScalar>());
    err  = WaitForGPU();CHKERRCUDA(err);
  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Thrust error: %s", ex);
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArray(win,&warray);CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* should do infinity norm in cuda */

PetscErrorCode VecNorm_SeqCUDA(Vec xin,NormType type,PetscReal *z)
{
  PetscErrorCode    ierr;
  PetscInt          n = xin->map->n;
  PetscBLASInt      one = 1, bn;
  const PetscScalar *xarray;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;
  hipError_t       err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(n,&bn);CHKERRQ(ierr);
  if (type == NORM_2 || type == NORM_FROBENIUS) {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXnrm2(cublasv2handle,bn,xarray,one,z);CHKERRCUBLAS(cberr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(PetscMax(2.0*n-1,0.0));CHKERRQ(ierr);
  } else if (type == NORM_INFINITY) {
    int  i;
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasIXamax(cublasv2handle,bn,xarray,one,&i);CHKERRCUBLAS(cberr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    if (bn) {
      PetscScalar zs;
      err = hipMemcpy(&zs,xarray+i-1,sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(err);
      *z = PetscAbsScalar(zs);
    } else *z = 0.0;
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  } else if (type == NORM_1) {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXasum(cublasv2handle,bn,xarray,one,z);CHKERRCUBLAS(cberr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(PetscMax(n-1.0,0.0));CHKERRQ(ierr);
  } else if (type == NORM_1_AND_2) {
    ierr = VecNorm_SeqCUDA(xin,NORM_1,z);CHKERRQ(ierr);
    ierr = VecNorm_SeqCUDA(xin,NORM_2,z+1);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecDotNorm2_SeqCUDA(Vec s, Vec t, PetscScalar *dp, PetscScalar *nm)
{
  PetscErrorCode    ierr;
  hipError_t       err;
  PetscReal         n=s->map->n;
  const PetscScalar *sarray,*tarray;

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayRead(s,&sarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(t,&tarray);CHKERRQ(ierr);
  ierr = VecDot_SeqCUDA(s,t,dp);CHKERRQ(ierr);
  ierr = VecDot_SeqCUDA(t,t,nm);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(s,&sarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(t,&tarray);CHKERRQ(ierr);
  err  = WaitForGPU();CHKERRCUDA(err);
  ierr = PetscLogGpuFlops(4.0*n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecDestroy_SeqCUDA(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;

  PetscFunctionBegin;
  if (v->spptr) {
    if (((Vec_CUDA*)v->spptr)->GPUarray_allocated) {
      err = hipFree(((Vec_CUDA*)v->spptr)->GPUarray_allocated);CHKERRCUDA(err);
      ((Vec_CUDA*)v->spptr)->GPUarray_allocated = NULL;
    }
    if (((Vec_CUDA*)v->spptr)->stream) {
      err = hipStreamDestroy(((Vec_CUDA*)v->spptr)->stream);CHKERRCUDA(err);
    }
  }
  ierr = VecDestroy_SeqCUDA_Private(v);CHKERRQ(ierr);
  ierr = PetscFree(v->spptr);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#if defined(PETSC_USE_COMPLEX)
struct conjugate
{
  __host__ __device__
    PetscScalar operator()(PetscScalar x)
    {
      return PetscConj(x);
    }
};
#endif

PetscErrorCode VecConjugate_SeqCUDA(Vec xin)
{
  PetscScalar                     *xarray;
  PetscErrorCode                  ierr;
#if defined(PETSC_USE_COMPLEX)
  PetscInt                        n = xin->map->n;
  thrust::device_ptr<PetscScalar> xptr;
  hipError_t                     err;
#endif

  PetscFunctionBegin;
  ierr = VecCUDAGetArray(xin,&xarray);CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  try {
    xptr = thrust::device_pointer_cast(xarray);
    thrust::transform(xptr,xptr+n,xptr,conjugate());
    err  = WaitForGPU();CHKERRCUDA(err);
  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Thrust error: %s", ex);
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
#endif
  ierr = VecCUDARestoreArray(xin,&xarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecGetLocalVector_SeqCUDA(Vec v,Vec w)
{
  PetscErrorCode ierr;
  hipError_t    err;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  PetscValidHeaderSpecific(w,VEC_CLASSID,2);
  PetscCheckTypeName(w,VECSEQCUDA);

  if (w->data) {
    if (((Vec_Seq*)w->data)->array_allocated) {
      if (w->pinned_memory) {
        ierr = PetscMallocSetCUDAHost();CHKERRQ(ierr);
      }
      ierr = PetscFree(((Vec_Seq*)w->data)->array_allocated);CHKERRQ(ierr);
      if (w->pinned_memory) {
        ierr = PetscMallocResetCUDAHost();CHKERRQ(ierr);
        w->pinned_memory = PETSC_FALSE;
      }
    }
    ((Vec_Seq*)w->data)->array = NULL;
    ((Vec_Seq*)w->data)->unplacedarray = NULL;
  }
  if (w->spptr) {
    if (((Vec_CUDA*)w->spptr)->GPUarray) {
      err = hipFree(((Vec_CUDA*)w->spptr)->GPUarray);CHKERRCUDA(err);
      ((Vec_CUDA*)w->spptr)->GPUarray = NULL;
    }
    if (((Vec_CUDA*)v->spptr)->stream) {
      err = hipStreamDestroy(((Vec_CUDA*)w->spptr)->stream);CHKERRCUDA(err);
    }
    ierr = PetscFree(w->spptr);CHKERRQ(ierr);
  }

  if (v->petscnative) {
    ierr = PetscFree(w->data);CHKERRQ(ierr);
    w->data = v->data;
    w->offloadmask = v->offloadmask;
    w->pinned_memory = v->pinned_memory;
    w->spptr = v->spptr;
    ierr = PetscObjectStateIncrease((PetscObject)w);CHKERRQ(ierr);
  } else {
    ierr = VecGetArray(v,&((Vec_Seq*)w->data)->array);CHKERRQ(ierr);
    w->offloadmask = PETSC_OFFLOAD_CPU;
    ierr = VecCUDAAllocateCheck(w);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecRestoreLocalVector_SeqCUDA(Vec v,Vec w)
{
  PetscErrorCode ierr;
  hipError_t    err;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  PetscValidHeaderSpecific(w,VEC_CLASSID,2);
  PetscCheckTypeName(w,VECSEQCUDA);

  if (v->petscnative) {
    v->data = w->data;
    v->offloadmask = w->offloadmask;
    v->pinned_memory = w->pinned_memory;
    v->spptr = w->spptr;
    ierr = VecCUDACopyFromGPU(v);CHKERRQ(ierr);
    ierr = PetscObjectStateIncrease((PetscObject)v);CHKERRQ(ierr);
    w->data = 0;
    w->offloadmask = PETSC_OFFLOAD_UNALLOCATED;
    w->spptr = 0;
  } else {
    ierr = VecRestoreArray(v,&((Vec_Seq*)w->data)->array);CHKERRQ(ierr);
    if ((Vec_CUDA*)w->spptr) {
      err = hipFree(((Vec_CUDA*)w->spptr)->GPUarray);CHKERRCUDA(err);
      ((Vec_CUDA*)w->spptr)->GPUarray = NULL;
      if (((Vec_CUDA*)v->spptr)->stream) {
        err = hipStreamDestroy(((Vec_CUDA*)w->spptr)->stream);CHKERRCUDA(err);
      }
      ierr = PetscFree(w->spptr);CHKERRQ(ierr);
    }
  }
  PetscFunctionReturn(0);
}
