#include "hip/hip_runtime.h"
#include <../src/vec/is/sf/impls/basic/sfpack.h>
#include <hip/hip_runtime.h>

/* Map a thread id to an index in root/leaf space through a series of 3D subdomains. See PetscSFPackOpt. */
__device__ static inline PetscInt MapTidToIndex(const PetscInt *opt,PetscInt tid)
{
  PetscInt        i,j,k,m,n,r;
  const PetscInt  *offset,*start,*dx,*dy,*X,*Y;

  n      = opt[0];
  offset = opt + 1;
  start  = opt + n + 2;
  dx     = opt + 2*n + 2;
  dy     = opt + 3*n + 2;
  X      = opt + 5*n + 2;
  Y      = opt + 6*n + 2;
  for (r=0; r<n; r++) {if (tid < offset[r+1]) break;}
  m = (tid - offset[r]);
  k = m/(dx[r]*dy[r]);
  j = (m - k*dx[r]*dy[r])/dx[r];
  i = m - k*dx[r]*dy[r] - j*dx[r];

  return (start[r] + k*X[r]*Y[r] + j*X[r] + i);
}

/*====================================================================================*/
/*  Templated CUDA kernels for pack/unpack. The Op can be regular or atomic           */
/*====================================================================================*/

/* Suppose user calls PetscSFReduce(sf,unit,...) and <unit> is an MPI data type made of 16 PetscReals, then
   <Type> is PetscReal, which is the primitive type we operate on.
   <bs>   is 16, which says <unit> contains 16 primitive types.
   <BS>   is 8, which is the maximal SIMD width we will try to vectorize operations on <unit>.
   <EQ>   is 0, which is (bs == BS ? 1 : 0)

  If instead, <unit> has 8 PetscReals, then bs=8, BS=8, EQ=1, rendering MBS below to a compile time constant.
  For the common case in VecScatter, bs=1, BS=1, EQ=1, MBS=1, the inner for-loops below will be totally unrolled.
*/
template<class Type,PetscInt BS,PetscInt EQ>
__global__ static void d_Pack(PetscInt bs,PetscInt count,PetscInt start,const PetscInt *opt,const PetscInt *idx,const Type *data,Type *buf)
{
  PetscInt        i,s,t,tid = blockIdx.x*blockDim.x + threadIdx.x;
  const PetscInt  grid_size = gridDim.x * blockDim.x;
  const PetscInt  M = (EQ) ? 1 : bs/BS; /* If EQ, then M=1 enables compiler's const-propagation */
  const PetscInt  MBS = M*BS;  /* MBS=bs. We turn MBS into a compile-time const when EQ=1. */

  for (; tid<count; tid += grid_size) {
    /* opt != NULL ==> idx == NULL, i.e., the indices have patterns but not contiguous;
       opt == NULL && idx == NULL ==> the indices are contiguous;
     */
    t = (opt? MapTidToIndex(opt,tid) : (idx? idx[tid] : start+tid))*MBS;
    s = tid*MBS;
    for (i=0; i<MBS; i++) buf[s+i] = data[t+i];
  }
}

template<class Type,class Op,PetscInt BS,PetscInt EQ>
__global__ static void d_UnpackAndOp(PetscInt bs,PetscInt count,PetscInt start,const PetscInt *opt,const PetscInt *idx,Type *data,const Type *buf)
{
  PetscInt        i,s,t,tid = blockIdx.x*blockDim.x + threadIdx.x;
  const PetscInt  grid_size = gridDim.x * blockDim.x;
  const PetscInt  M = (EQ) ? 1 : bs/BS, MBS = M*BS;
  Op              op;

  for (; tid<count; tid += grid_size) {
    t = (opt? MapTidToIndex(opt,tid) : (idx? idx[tid] : start+tid))*MBS;
    s = tid*MBS;
    for (i=0; i<MBS; i++) op(data[t+i],buf[s+i]);
  }
}

template<class Type,class Op,PetscInt BS,PetscInt EQ>
__global__ static void d_FetchAndOp(PetscInt bs,PetscInt count,PetscInt rootstart,const PetscInt *rootopt,const PetscInt *rootidx,Type *rootdata,Type *leafbuf)
{
  PetscInt        i,l,r,tid = blockIdx.x*blockDim.x + threadIdx.x;
  const PetscInt  grid_size = gridDim.x * blockDim.x;
  const PetscInt  M = (EQ) ? 1 : bs/BS, MBS = M*BS;
  Op              op;

  for (; tid<count; tid += grid_size) {
    r = (rootopt? MapTidToIndex(rootopt,tid) : (rootidx? rootidx[tid] : rootstart+tid))*MBS;
    l = tid*MBS;
    for (i=0; i<MBS; i++) leafbuf[l+i] = op(rootdata[r+i],leafbuf[l+i]);
  }
}

template<class Type,class Op,PetscInt BS,PetscInt EQ>
__global__ static void d_ScatterAndOp(PetscInt bs,PetscInt count,PetscInt srcx,PetscInt srcy,PetscInt srcX,PetscInt srcY,PetscInt srcStart,const PetscInt* srcIdx,const Type *src,PetscInt dstx,PetscInt dsty,PetscInt dstX,PetscInt dstY,PetscInt dstStart,const PetscInt *dstIdx,Type *dst)
{
  PetscInt        i,j,k,s,t,tid = blockIdx.x*blockDim.x + threadIdx.x;
  const PetscInt  grid_size = gridDim.x * blockDim.x;
  const PetscInt  M = (EQ) ? 1 : bs/BS, MBS = M*BS;
  Op              op;

  for (; tid<count; tid += grid_size) {
    if (!srcIdx) { /* src is either contiguous or 3D */
      k = tid/(srcx*srcy);
      j = (tid - k*srcx*srcy)/srcx;
      i = tid - k*srcx*srcy - j*srcx;
      s = srcStart + k*srcX*srcY + j*srcX + i;
    } else {
      s = srcIdx[tid];
    }

    if (!dstIdx) { /* dst is either contiguous or 3D */
      k = tid/(dstx*dsty);
      j = (tid - k*dstx*dsty)/dstx;
      i = tid - k*dstx*dsty - j*dstx;
      t = dstStart + k*dstX*dstY + j*dstX + i;
    } else {
      t = dstIdx[tid];
    }

    s *= MBS;
    t *= MBS;
    for (i=0; i<MBS; i++) op(dst[t+i],src[s+i]);
  }
}

template<class Type,class Op,PetscInt BS,PetscInt EQ>
__global__ static void d_FetchAndOpLocal(PetscInt bs,PetscInt count,PetscInt rootstart,const PetscInt *rootopt,const PetscInt *rootidx,Type *rootdata,PetscInt leafstart,const PetscInt *leafopt,const PetscInt *leafidx,const Type *leafdata,Type *leafupdate)
{
  PetscInt        i,l,r,tid = blockIdx.x*blockDim.x + threadIdx.x;
  const PetscInt  grid_size = gridDim.x * blockDim.x;
  const PetscInt  M = (EQ) ? 1 : bs/BS, MBS = M*BS;
  Op              op;

  for (; tid<count; tid += grid_size) {
    r = (rootopt? MapTidToIndex(rootopt,tid) : (rootidx? rootidx[tid] : rootstart+tid))*MBS;
    l = (leafopt? MapTidToIndex(leafopt,tid) : (leafidx? leafidx[tid] : leafstart+tid))*MBS;
    for (i=0; i<MBS; i++) leafupdate[l+i] = op(rootdata[r+i],leafdata[l+i]);
  }
}

/*====================================================================================*/
/*                             Regular operations on device                           */
/*====================================================================================*/
template<typename Type> struct Insert {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = y;             return old;}};
template<typename Type> struct Add    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x += y;             return old;}};
template<typename Type> struct Mult   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x *= y;             return old;}};
template<typename Type> struct Min    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = PetscMin(x,y); return old;}};
template<typename Type> struct Max    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = PetscMax(x,y); return old;}};
template<typename Type> struct LAND   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x && y;        return old;}};
template<typename Type> struct LOR    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x || y;        return old;}};
template<typename Type> struct LXOR   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = !x != !y;      return old;}};
template<typename Type> struct BAND   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x & y;         return old;}};
template<typename Type> struct BOR    {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x | y;         return old;}};
template<typename Type> struct BXOR   {__device__ Type operator() (Type& x,Type y) const {Type old = x; x  = x ^ y;         return old;}};
template<typename Type> struct Minloc {
  __device__ Type operator() (Type& x,Type y) const {
    Type old = x;
    if (y.a < x.a) x = y;
    else if (y.a == x.a) x.b = min(x.b,y.b);
    return old;
  }
};
template<typename Type> struct Maxloc {
  __device__ Type operator() (Type& x,Type y) const {
    Type old = x;
    if (y.a > x.a) x = y;
    else if (y.a == x.a) x.b = min(x.b,y.b); /* See MPI MAXLOC */
    return old;
  }
};

/*====================================================================================*/
/*                             Atomic operations on device                            */
/*====================================================================================*/

/*
  Atomic Insert (exchange) operations

  CUDA C Programming Guide V10.1 Chapter B.12.1.3:

  int atomicExch(int* address, int val);
  unsigned int atomicExch(unsigned int* address, unsigned int val);
  unsigned long long int atomicExch(unsigned long long int* address, unsigned long long int val);
  float atomicExch(float* address, float val);

  reads the 32-bit or 64-bit word old located at the address address in global or shared
  memory and stores val back to memory at the same address. These two operations are
  performed in one atomic transaction. The function returns old.

  PETSc notes:

  It may be useful in PetscSFFetchAndOp with op = MPIU_REPLACE.

  VecScatter with multiple entries scattered to the same location using INSERT_VALUES does not need
  atomic insertion, since it does not need the old value. A 32-bit or 64-bit store instruction should
  be atomic itself.

  With bs>1 and a unit > 64 bits, the current element-wise atomic approach can not guarantee the whole
  insertion is atomic. Hope no user codes rely on that.
*/

#if defined(PETSC_USE_REAL_DOUBLE)
__device__ static double atomicExch(double* address,double val) {return __longlong_as_double(atomicExch((unsigned long long int*)address,__double_as_longlong(val)));}
#endif

#if defined(PETSC_USE_64BIT_INDICES)
__device__ static PetscInt atomicExch(PetscInt* address,PetscInt val) {return (PetscInt)(atomicExch((unsigned long long int*)address,(unsigned long long int)val));}
#endif

template<typename Type> struct AtomicInsert {__device__ Type operator() (Type& x,Type y) const {return atomicExch(&x,y);}};

/*
  Atomic add operations

  CUDA C Programming Guide V10.1 Chapter B.12.1.1:

  int atomicAdd(int* address, int val);
  unsigned int atomicAdd(unsigned int* address,unsigned int val);
  unsigned long long int atomicAdd(unsigned long long int* address,unsigned long long int val);
  float atomicAdd(float* address, float val);
  double atomicAdd(double* address, double val);
  __half2 atomicAdd(__half2 *address, __half2 val);
  __half atomicAdd(__half *address, __half val);

  reads the 16-bit, 32-bit or 64-bit word old located at the address address in global or shared memory, computes (old + val),
  and stores the result back to memory at the same address. These three operations are performed in one atomic transaction. The
  function returns old.

  The 32-bit floating-point version of atomicAdd() is only supported by devices of compute capability 2.x and higher.
  The 64-bit floating-point version of atomicAdd() is only supported by devices of compute capability 6.x and higher.
  The 32-bit __half2 floating-point version of atomicAdd() is only supported by devices of compute capability 6.x and
  higher. The atomicity of the __half2 add operation is guaranteed separately for each of the two __half elements;
  the entire __half2 is not guaranteed to be atomic as a single 32-bit access.
  The 16-bit __half floating-point version of atomicAdd() is only supported by devices of compute capability 7.x and higher.
*/

#if defined(PETSC_USE_64BIT_INDICES)
__device__ static PetscInt atomicAdd(PetscInt* address,PetscInt val) {return (PetscInt)atomicAdd((unsigned long long int*)address,(unsigned long long int)val);}
#endif

template<typename Type> struct AtomicAdd {__device__ Type operator() (Type& x,Type y) const {return atomicAdd(&x,y);}};

template<> struct AtomicAdd<double> {
  __device__ double operator() (double& x,double y) const {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 600)
    return atomicAdd(&x,y);
#else
    double                 *address = &x, val = y;
    unsigned long long int *address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
      assumed = old;
      old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
      /* Note: uses integer comparison to avoid hang in case of NaN (since NaN !=NaN) */
    } while (assumed != old);
    return __longlong_as_double(old);
#endif
  }
};

template<> struct AtomicAdd<float> {
  __device__ float operator() (float& x,float y) const {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 200)
    return atomicAdd(&x,y);
#else
    float *address = &x, val = y;
    int   *address_as_int = (int*)address;
    int   old = *address_as_int, assumed;
    do {
      assumed = old;
      old     = atomicCAS(address_as_int, assumed, __float_as_int(val + __int_as_float(assumed)));
      /* Note: uses integer comparison to avoid hang in case of NaN (since NaN !=NaN) */
    } while (assumed != old);
    return __int_as_float(old);
#endif
  }
};

template<> struct AtomicAdd<PetscComplex> {
 __device__ PetscComplex operator() (PetscComplex& x,PetscComplex y) const {
  PetscComplex         old, *z = &old;
  PetscReal            *xp = (PetscReal*)&x,*yp = (PetscReal*)&y;
  AtomicAdd<PetscReal> op;
  z[0] = op(xp[0],yp[0]);
  z[1] = op(xp[1],yp[1]);
  return old; /* The returned value may not be atomic. It can be mix of two ops. Caller should discard it. */
 }
};

/*
  Atomic Mult operations:

  CUDA has no atomicMult at all, so we build our own with atomicCAS
 */
#if defined(PETSC_USE_REAL_DOUBLE)
__device__ static double atomicMult(double* address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    /* Other threads can access and modify value of *address_as_ull after the read above and before the write below */
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(val*__longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#elif defined(PETSC_USE_REAL_SINGLE)
__device__ static float atomicMult(float* address,float val)
{
  int *address_as_int = (int*)(address);
  int old = *address_as_int, assumed;
  do {
    assumed  = old;
    old      = atomicCAS(address_as_int, assumed, __float_as_int(val*__int_as_float(assumed)));
  } while (assumed != old);
  return __int_as_float(old);
}
#endif

__device__ static int atomicMult(int* address,int val)
{
  int *address_as_int = (int*)(address);
  int old = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, val*assumed);
  } while (assumed != old);
  return (int)old;
}

#if defined(PETSC_USE_64BIT_INDICES)
__device__ static int atomicMult(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(val*(PetscInt)assumed));
  } while (assumed != old);
  return (PetscInt)old;
}
#endif

template<typename Type> struct AtomicMult {__device__ Type operator() (Type& x,Type y) const {return atomicMult(&x,y);}};

/*
  Atomic Min/Max operations

  CUDA C Programming Guide V10.1 Chapter B.12.1.4~5:

  int atomicMin(int* address, int val);
  unsigned int atomicMin(unsigned int* address,unsigned int val);
  unsigned long long int atomicMin(unsigned long long int* address,unsigned long long int val);

  reads the 32-bit or 64-bit word old located at the address address in global or shared
  memory, computes the minimum of old and val, and stores the result back to memory
  at the same address. These three operations are performed in one atomic transaction.
  The function returns old.
  The 64-bit version of atomicMin() is only supported by devices of compute capability 3.5 and higher.

  atomicMax() is similar.
 */

#if defined(PETSC_USE_REAL_DOUBLE)
__device__ static double atomicMin(double* address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(PetscMin(val,__longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}

__device__ static double atomicMax(double* address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed  = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(PetscMax(val,__longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#elif defined(PETSC_USE_REAL_SINGLE)
__device__ static float atomicMin(float* address,float val)
{
  int *address_as_int = (int*)(address);
  int old = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, __float_as_int(PetscMin(val,__int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__device__ static float atomicMax(float* address,float val)
{
  int *address_as_int = (int*)(address);
  int old = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, __float_as_int(PetscMax(val,__int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}
#endif

/*
  atomicMin/Max(long long *, long long) are not in Nvidia's documentation. But on OLCF Summit we found
  atomicMin/Max/And/Or/Xor(long long *, long long) in /sw/summit/cuda/10.1.243/include/sm_32_atomic_functions.h.
  This causes compilation errors with pgi compilers and 64-bit indices:
      error: function "atomicMin(long long *, long long)" has already been defined

  So we add extra conditions defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 320)
*/
#if defined(PETSC_USE_64BIT_INDICES) && defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 320)
__device__ static PetscInt atomicMin(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(PetscMin(val,(PetscInt)assumed)));
  } while (assumed != old);
  return (PetscInt)old;
}

__device__ static PetscInt atomicMax(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(PetscMax(val,(PetscInt)assumed)));
  } while (assumed != old);
  return (PetscInt)old;
}
#endif

template<typename Type> struct AtomicMin {__device__ Type operator() (Type& x,Type y) const {return atomicMin(&x,y);}};
template<typename Type> struct AtomicMax {__device__ Type operator() (Type& x,Type y) const {return atomicMax(&x,y);}};

/*
  Atomic bitwise operations

  CUDA C Programming Guide V10.1 Chapter B.12.2.1 ~ B.12.2.3:

  int atomicAnd(int* address, int val);
  unsigned int atomicAnd(unsigned int* address,unsigned int val);
  unsigned long long int atomicAnd(unsigned long long int* address,unsigned long long int val);

  reads the 32-bit or 64-bit word old located at the address address in global or shared
  memory, computes (old & val), and stores the result back to memory at the same
  address. These three operations are performed in one atomic transaction.
  The function returns old.

  The 64-bit version of atomicAnd() is only supported by devices of compute capability 3.5 and higher.

  atomicOr() and atomicXor are similar.
*/

#if defined(PETSC_USE_64BIT_INDICES)
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 320) /* Why 320? see comments at atomicMin(PetscInt* address,PetscInt val) */
__device__ static PetscInt atomicAnd(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(val & (PetscInt)assumed));
  } while (assumed != old);
  return (PetscInt)old;
}
__device__ static PetscInt atomicOr(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(val | (PetscInt)assumed));
  } while (assumed != old);
  return (PetscInt)old;
}

__device__ static PetscInt atomicXor(PetscInt* address,PetscInt val)
{
  unsigned long long int *address_as_ull = (unsigned long long int*)(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(val ^ (PetscInt)assumed));
  } while (assumed != old);
  return (PetscInt)old;
}
#else
/*
 See also comments at atomicMin(PetscInt* address,PetscInt val)
__device__ static PetscInt atomicAnd(PetscInt* address,PetscInt val) {return (PetscInt)atomicAnd((unsigned long long int*)address,(unsigned long long int)val);}
__device__ static PetscInt atomicOr (PetscInt* address,PetscInt val) {return (PetscInt)atomicOr ((unsigned long long int*)address,(unsigned long long int)val);}
__device__ static PetscInt atomicXor(PetscInt* address,PetscInt val) {return (PetscInt)atomicXor((unsigned long long int*)address,(unsigned long long int)val);}
*/
#endif
#endif

template<typename Type> struct AtomicBAND {__device__ Type operator() (Type& x,Type y) const {return atomicAnd(&x,y);}};
template<typename Type> struct AtomicBOR  {__device__ Type operator() (Type& x,Type y) const {return atomicOr (&x,y);}};
template<typename Type> struct AtomicBXOR {__device__ Type operator() (Type& x,Type y) const {return atomicXor(&x,y);}};

/*
  Atomic logical operations:

  CUDA has no atomic logical operations at all. We support them on integer types.
*/

/* A template without definition makes any instantiation not using given specializations erroneous at compile time,
   which is what we want since we only support 32-bit and 64-bit integers.
 */
template<typename Type,class Op,int size/* sizeof(Type) */> struct AtomicLogical;

template<typename Type,class Op>
struct AtomicLogical<Type,Op,4> {
  __device__ Type operator()(Type& x,Type y) const {
    int *address_as_int = (int*)(&x);
    int old = *address_as_int, assumed;
    Op op;
    do {
      assumed = old;
      old     = atomicCAS(address_as_int, assumed, (int)(op((Type)assumed,y)));
    } while (assumed != old);
    return (Type)old;
  }
};

template<typename Type,class Op>
struct AtomicLogical<Type,Op,8> {
  __device__ Type operator()(Type& x,Type y) const {
    unsigned long long int *address_as_ull = (unsigned long long int*)(&x);
    unsigned long long int old = *address_as_ull, assumed;
    Op op;
    do {
      assumed = old;
      old     = atomicCAS(address_as_ull, assumed, (unsigned long long int)(op((Type)assumed,y)));
    } while (assumed != old);
    return (Type)old;
  }
};

/* Note land/lor/lxor below are different from LAND etc above. Here we pass arguments by value and return result of ops (not old value) */
template<typename Type> struct land {__device__ Type operator()(Type x, Type y) {return x && y;}};
template<typename Type> struct lor  {__device__ Type operator()(Type x, Type y) {return x || y;}};
template<typename Type> struct lxor {__device__ Type operator()(Type x, Type y) {return (!x != !y);}};

template<typename Type> struct AtomicLAND {__device__ Type operator()(Type& x,Type y) const {AtomicLogical<Type,land<Type>,sizeof(Type)> op; return op(x,y);}};
template<typename Type> struct AtomicLOR  {__device__ Type operator()(Type& x,Type y) const {AtomicLogical<Type,lor<Type> ,sizeof(Type)> op; return op(x,y);}};
template<typename Type> struct AtomicLXOR {__device__ Type operator()(Type& x,Type y) const {AtomicLogical<Type,lxor<Type>,sizeof(Type)> op; return op(x,y);}};

/*====================================================================================*/
/*  Wrapper functions of cuda kernels. Function pointers are stored in 'link'         */
/*====================================================================================*/
template<typename Type,PetscInt BS,PetscInt EQ>
static PetscErrorCode Pack(PetscSFLink link,PetscInt count,PetscInt start,PetscSFPackOpt opt,const PetscInt *idx,const void *data,void *buf)
{
  hipError_t        cerr;
  PetscInt           nthreads=256;
  PetscInt           nblocks=(count+nthreads-1)/nthreads;
  const PetscInt     *iarray=opt ? opt->array : NULL;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(0);
  nblocks = PetscMin(nblocks,link->maxResidentThreadsPerGPU/nthreads);
  d_Pack<Type,BS,EQ><<<nblocks,nthreads,0,link->stream>>>(link->bs,count,start,iarray,idx,(const Type*)data,(Type*)buf);
  cerr = hipGetLastError();CHKERRCUDA(cerr);
  PetscFunctionReturn(0);
}

template<typename Type,class Op,PetscInt BS,PetscInt EQ>
static PetscErrorCode UnpackAndOp(PetscSFLink link,PetscInt count,PetscInt start,PetscSFPackOpt opt,const PetscInt *idx,void *data,const void *buf)
{
  hipError_t        cerr;
  PetscInt           nthreads=256;
  PetscInt           nblocks=(count+nthreads-1)/nthreads;
  const PetscInt     *iarray=opt ? opt->array : NULL;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(0);
  nblocks = PetscMin(nblocks,link->maxResidentThreadsPerGPU/nthreads);
  d_UnpackAndOp<Type,Op,BS,EQ><<<nblocks,nthreads,0,link->stream>>>(link->bs,count,start,iarray,idx,(Type*)data,(const Type*)buf);
  cerr = hipGetLastError();CHKERRCUDA(cerr);
  PetscFunctionReturn(0);
}

template<typename Type,class Op,PetscInt BS,PetscInt EQ>
static PetscErrorCode FetchAndOp(PetscSFLink link,PetscInt count,PetscInt start,PetscSFPackOpt opt,const PetscInt *idx,void *data,void *buf)
{
  hipError_t        cerr;
  PetscInt           nthreads=256;
  PetscInt           nblocks=(count+nthreads-1)/nthreads;
  const PetscInt     *iarray=opt ? opt->array : NULL;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(0);
  nblocks = PetscMin(nblocks,link->maxResidentThreadsPerGPU/nthreads);
  d_FetchAndOp<Type,Op,BS,EQ><<<nblocks,nthreads,0,link->stream>>>(link->bs,count,start,iarray,idx,(Type*)data,(Type*)buf);
  cerr = hipGetLastError();CHKERRCUDA(cerr);
  PetscFunctionReturn(0);
}

template<typename Type,class Op,PetscInt BS,PetscInt EQ>
static PetscErrorCode ScatterAndOp(PetscSFLink link,PetscInt count,PetscInt srcStart,PetscSFPackOpt srcOpt,const PetscInt *srcIdx,const void *src,PetscInt dstStart,PetscSFPackOpt dstOpt,const PetscInt *dstIdx,void *dst)
{
  hipError_t        cerr;
  PetscInt           nthreads=256;
  PetscInt           nblocks=(count+nthreads-1)/nthreads;
  PetscInt           srcx=0,srcy=0,srcX=0,srcY=0,dstx=0,dsty=0,dstX=0,dstY=0;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(0);
  nblocks = PetscMin(nblocks,link->maxResidentThreadsPerGPU/nthreads);

  /* The 3D shape of source subdomain may be different than that of the destination, which makes it difficult to use CUDA 3D grid and block */
  if (srcOpt)       {srcx = srcOpt->dx[0]; srcy = srcOpt->dy[0]; srcX = srcOpt->X[0]; srcY = srcOpt->Y[0]; srcStart = srcOpt->start[0]; srcIdx = NULL;}
  else if (!srcIdx) {srcx = srcX = count; srcy = srcY = 1;}

  if (dstOpt)       {dstx = dstOpt->dx[0]; dsty = dstOpt->dy[0]; dstX = dstOpt->X[0]; dstY = dstOpt->Y[0]; dstStart = dstOpt->start[0]; dstIdx = NULL;}
  else if (!dstIdx) {dstx = dstX = count; dsty = dstY = 1;}

  d_ScatterAndOp<Type,Op,BS,EQ><<<nblocks,nthreads,0,link->stream>>>(link->bs,count,srcx,srcy,srcX,srcY,srcStart,srcIdx,(const Type*)src,dstx,dsty,dstX,dstY,dstStart,dstIdx,(Type*)dst);
  cerr = hipGetLastError();CHKERRCUDA(cerr);
  PetscFunctionReturn(0);
}

/* Specialization for Insert since we may use hipMemcpyAsync */
template<typename Type,PetscInt BS,PetscInt EQ>
static PetscErrorCode ScatterAndInsert(PetscSFLink link,PetscInt count,PetscInt srcStart,PetscSFPackOpt srcOpt,const PetscInt *srcIdx,const void *src,PetscInt dstStart,PetscSFPackOpt dstOpt,const PetscInt *dstIdx,void *dst)
{
  PetscErrorCode    ierr;
  hipError_t       cerr;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(0);
  /*src and dst are contiguous */
  if ((!srcOpt && !srcIdx) && (!dstOpt && !dstIdx) && src != dst) {
    cerr = hipMemcpyAsync((Type*)dst+dstStart*link->bs,(const Type*)src+srcStart*link->bs,count*link->unitbytes,hipMemcpyDeviceToDevice,link->stream);CHKERRCUDA(cerr);
  } else {
    ierr = ScatterAndOp<Type,Insert<Type>,BS,EQ>(link,count,srcStart,srcOpt,srcIdx,src,dstStart,dstOpt,dstIdx,dst);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

template<typename Type,class Op,PetscInt BS,PetscInt EQ>
static PetscErrorCode FetchAndOpLocal(PetscSFLink link,PetscInt count,PetscInt rootstart,PetscSFPackOpt rootopt,const PetscInt *rootidx,void *rootdata,PetscInt leafstart,PetscSFPackOpt leafopt,const PetscInt *leafidx,const void *leafdata,void *leafupdate)
{
  hipError_t       cerr;
  PetscInt          nthreads=256;
  PetscInt          nblocks=(count+nthreads-1)/nthreads;
  const PetscInt    *rarray = rootopt ? rootopt->array : NULL;
  const PetscInt    *larray = leafopt ? leafopt->array : NULL;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(0);
  nblocks = PetscMin(nblocks,link->maxResidentThreadsPerGPU/nthreads);
  d_FetchAndOpLocal<Type,Op,BS,EQ><<<nblocks,nthreads,0,link->stream>>>(link->bs,count,rootstart,rarray,rootidx,(Type*)rootdata,leafstart,larray,leafidx,(const Type*)leafdata,(Type*)leafupdate);
  cerr = hipGetLastError();CHKERRCUDA(cerr);
  PetscFunctionReturn(0);
}

/*====================================================================================*/
/*  Init various types and instantiate pack/unpack function pointers                  */
/*====================================================================================*/
template<typename Type,PetscInt BS,PetscInt EQ>
static void PackInit_RealType(PetscSFLink link)
{
  /* Pack/unpack for remote communication */
  link->d_Pack              = Pack<Type,BS,EQ>;
  link->d_UnpackAndInsert   = UnpackAndOp     <Type,Insert<Type>      ,BS,EQ>;
  link->d_UnpackAndAdd      = UnpackAndOp     <Type,Add<Type>         ,BS,EQ>;
  link->d_UnpackAndMult     = UnpackAndOp     <Type,Mult<Type>        ,BS,EQ>;
  link->d_UnpackAndMin      = UnpackAndOp     <Type,Min<Type>         ,BS,EQ>;
  link->d_UnpackAndMax      = UnpackAndOp     <Type,Max<Type>         ,BS,EQ>;
  link->d_FetchAndAdd       = FetchAndOp      <Type,Add<Type>         ,BS,EQ>;

  /* Scatter for local communication */
  link->d_ScatterAndInsert  = ScatterAndInsert<Type                   ,BS,EQ>; /* Has special optimizations */
  link->d_ScatterAndAdd     = ScatterAndOp    <Type,Add<Type>         ,BS,EQ>;
  link->d_ScatterAndMult    = ScatterAndOp    <Type,Mult<Type>        ,BS,EQ>;
  link->d_ScatterAndMin     = ScatterAndOp    <Type,Min<Type>         ,BS,EQ>;
  link->d_ScatterAndMax     = ScatterAndOp    <Type,Max<Type>         ,BS,EQ>;
  link->d_FetchAndAddLocal  = FetchAndOpLocal <Type,Add <Type>        ,BS,EQ>;

  /* Atomic versions when there are data-race possibilities */
  link->da_UnpackAndInsert  = UnpackAndOp     <Type,AtomicInsert<Type>,BS,EQ>;
  link->da_UnpackAndAdd     = UnpackAndOp     <Type,AtomicAdd<Type>   ,BS,EQ>;
  link->da_UnpackAndMult    = UnpackAndOp     <Type,AtomicMult<Type>  ,BS,EQ>;
  link->da_UnpackAndMin     = UnpackAndOp     <Type,AtomicMin<Type>   ,BS,EQ>;
  link->da_UnpackAndMax     = UnpackAndOp     <Type,AtomicMax<Type>   ,BS,EQ>;
  link->da_FetchAndAdd      = FetchAndOp      <Type,AtomicAdd<Type>   ,BS,EQ>;

  link->da_ScatterAndInsert = ScatterAndOp    <Type,AtomicInsert<Type>,BS,EQ>;
  link->da_ScatterAndAdd    = ScatterAndOp    <Type,AtomicAdd<Type>   ,BS,EQ>;
  link->da_ScatterAndMult   = ScatterAndOp    <Type,AtomicMult<Type>  ,BS,EQ>;
  link->da_ScatterAndMin    = ScatterAndOp    <Type,AtomicMin<Type>   ,BS,EQ>;
  link->da_ScatterAndMax    = ScatterAndOp    <Type,AtomicMax<Type>   ,BS,EQ>;
  link->da_FetchAndAddLocal = FetchAndOpLocal <Type,AtomicAdd<Type>   ,BS,EQ>;
}

/* Have this templated class to specialize for char integers */
template<typename Type,PetscInt BS,PetscInt EQ,PetscInt size/*sizeof(Type)*/>
struct PackInit_IntegerType_Atomic {
  static void Init(PetscSFLink link) {
    link->da_UnpackAndInsert  = UnpackAndOp<Type,AtomicInsert<Type>,BS,EQ>;
    link->da_UnpackAndAdd     = UnpackAndOp<Type,AtomicAdd<Type>   ,BS,EQ>;
    link->da_UnpackAndMult    = UnpackAndOp<Type,AtomicMult<Type>  ,BS,EQ>;
    link->da_UnpackAndMin     = UnpackAndOp<Type,AtomicMin<Type>   ,BS,EQ>;
    link->da_UnpackAndMax     = UnpackAndOp<Type,AtomicMax<Type>   ,BS,EQ>;
    link->da_UnpackAndLAND    = UnpackAndOp<Type,AtomicLAND<Type>  ,BS,EQ>;
    link->da_UnpackAndLOR     = UnpackAndOp<Type,AtomicLOR<Type>   ,BS,EQ>;
    link->da_UnpackAndLXOR    = UnpackAndOp<Type,AtomicLXOR<Type>  ,BS,EQ>;
    link->da_UnpackAndBAND    = UnpackAndOp<Type,AtomicBAND<Type>  ,BS,EQ>;
    link->da_UnpackAndBOR     = UnpackAndOp<Type,AtomicBOR<Type>   ,BS,EQ>;
    link->da_UnpackAndBXOR    = UnpackAndOp<Type,AtomicBXOR<Type>  ,BS,EQ>;
    link->da_FetchAndAdd      = FetchAndOp <Type,AtomicAdd<Type>   ,BS,EQ>;

    link->da_ScatterAndInsert = ScatterAndOp<Type,AtomicInsert<Type>,BS,EQ>;
    link->da_ScatterAndAdd    = ScatterAndOp<Type,AtomicAdd<Type>   ,BS,EQ>;
    link->da_ScatterAndMult   = ScatterAndOp<Type,AtomicMult<Type>  ,BS,EQ>;
    link->da_ScatterAndMin    = ScatterAndOp<Type,AtomicMin<Type>   ,BS,EQ>;
    link->da_ScatterAndMax    = ScatterAndOp<Type,AtomicMax<Type>   ,BS,EQ>;
    link->da_ScatterAndLAND   = ScatterAndOp<Type,AtomicLAND<Type>  ,BS,EQ>;
    link->da_ScatterAndLOR    = ScatterAndOp<Type,AtomicLOR<Type>   ,BS,EQ>;
    link->da_ScatterAndLXOR   = ScatterAndOp<Type,AtomicLXOR<Type>  ,BS,EQ>;
    link->da_ScatterAndBAND   = ScatterAndOp<Type,AtomicBAND<Type>  ,BS,EQ>;
    link->da_ScatterAndBOR    = ScatterAndOp<Type,AtomicBOR<Type>   ,BS,EQ>;
    link->da_ScatterAndBXOR   = ScatterAndOp<Type,AtomicBXOR<Type>  ,BS,EQ>;
    link->da_FetchAndAddLocal = FetchAndOpLocal<Type,AtomicAdd<Type>,BS,EQ>;
  }
};

/* CUDA does not support atomics on chars. It is TBD in PETSc. */
template<typename Type,PetscInt BS,PetscInt EQ>
struct PackInit_IntegerType_Atomic<Type,BS,EQ,1> {
  static void Init(PetscSFLink link) {/* Nothing to leave function pointers NULL */}
};

template<typename Type,PetscInt BS,PetscInt EQ>
static void PackInit_IntegerType(PetscSFLink link)
{
  link->d_Pack            = Pack<Type,BS,EQ>;
  link->d_UnpackAndInsert = UnpackAndOp<Type,Insert<Type>,BS,EQ>;
  link->d_UnpackAndAdd    = UnpackAndOp<Type,Add<Type>   ,BS,EQ>;
  link->d_UnpackAndMult   = UnpackAndOp<Type,Mult<Type>  ,BS,EQ>;
  link->d_UnpackAndMin    = UnpackAndOp<Type,Min<Type>   ,BS,EQ>;
  link->d_UnpackAndMax    = UnpackAndOp<Type,Max<Type>   ,BS,EQ>;
  link->d_UnpackAndLAND   = UnpackAndOp<Type,LAND<Type>  ,BS,EQ>;
  link->d_UnpackAndLOR    = UnpackAndOp<Type,LOR<Type>   ,BS,EQ>;
  link->d_UnpackAndLXOR   = UnpackAndOp<Type,LXOR<Type>  ,BS,EQ>;
  link->d_UnpackAndBAND   = UnpackAndOp<Type,BAND<Type>  ,BS,EQ>;
  link->d_UnpackAndBOR    = UnpackAndOp<Type,BOR<Type>   ,BS,EQ>;
  link->d_UnpackAndBXOR   = UnpackAndOp<Type,BXOR<Type>  ,BS,EQ>;
  link->d_FetchAndAdd     = FetchAndOp <Type,Add<Type>   ,BS,EQ>;

  link->d_ScatterAndInsert = ScatterAndInsert<Type,BS,EQ>;
  link->d_ScatterAndAdd    = ScatterAndOp<Type,Add<Type>   ,BS,EQ>;
  link->d_ScatterAndMult   = ScatterAndOp<Type,Mult<Type>  ,BS,EQ>;
  link->d_ScatterAndMin    = ScatterAndOp<Type,Min<Type>   ,BS,EQ>;
  link->d_ScatterAndMax    = ScatterAndOp<Type,Max<Type>   ,BS,EQ>;
  link->d_ScatterAndLAND   = ScatterAndOp<Type,LAND<Type>  ,BS,EQ>;
  link->d_ScatterAndLOR    = ScatterAndOp<Type,LOR<Type>   ,BS,EQ>;
  link->d_ScatterAndLXOR   = ScatterAndOp<Type,LXOR<Type>  ,BS,EQ>;
  link->d_ScatterAndBAND   = ScatterAndOp<Type,BAND<Type>  ,BS,EQ>;
  link->d_ScatterAndBOR    = ScatterAndOp<Type,BOR<Type>   ,BS,EQ>;
  link->d_ScatterAndBXOR   = ScatterAndOp<Type,BXOR<Type>  ,BS,EQ>;
  link->d_FetchAndAddLocal = FetchAndOpLocal<Type,Add<Type>,BS,EQ>;
  PackInit_IntegerType_Atomic<Type,BS,EQ,sizeof(Type)>::Init(link);
}

#if defined(PETSC_HAVE_COMPLEX)
template<typename Type,PetscInt BS,PetscInt EQ>
static void PackInit_ComplexType(PetscSFLink link)
{
  link->d_Pack             = Pack<Type,BS,EQ>;
  link->d_UnpackAndInsert  = UnpackAndOp<Type,Insert<Type>,BS,EQ>;
  link->d_UnpackAndAdd     = UnpackAndOp<Type,Add<Type>   ,BS,EQ>;
  link->d_UnpackAndMult    = UnpackAndOp<Type,Mult<Type>  ,BS,EQ>;
  link->d_FetchAndAdd      = FetchAndOp <Type,Add<Type>   ,BS,EQ>;

  link->d_ScatterAndInsert = ScatterAndInsert<Type,BS,EQ>;
  link->d_ScatterAndAdd    = ScatterAndOp<Type,Add<Type>   ,BS,EQ>;
  link->d_ScatterAndMult   = ScatterAndOp<Type,Mult<Type>  ,BS,EQ>;
  link->d_FetchAndAddLocal = FetchAndOpLocal<Type,Add<Type>,BS,EQ>;

  link->da_UnpackAndAdd    = UnpackAndOp<Type,AtomicAdd<Type>,BS,EQ>;
  link->da_UnpackAndMult   = NULL; /* Not implemented yet */
  link->da_FetchAndAdd     = NULL; /* Return value of atomicAdd on complex is not atomic */
  link->da_ScatterAndAdd   = ScatterAndOp<Type,AtomicAdd<Type>,BS,EQ>;
}
#endif

typedef signed char                      SignedChar;
typedef unsigned char                    UnsignedChar;
typedef struct {int a;      int b;     } PairInt;
typedef struct {PetscInt a; PetscInt b;} PairPetscInt;

template<typename Type>
static void PackInit_PairType(PetscSFLink link)
{
  link->d_Pack            = Pack<Type,1,1>;
  link->d_UnpackAndInsert = UnpackAndOp<Type,Insert<Type>,1,1>;
  link->d_UnpackAndMaxloc = UnpackAndOp<Type,Maxloc<Type>,1,1>;
  link->d_UnpackAndMinloc = UnpackAndOp<Type,Minloc<Type>,1,1>;

  link->d_ScatterAndInsert = ScatterAndOp<Type,Insert<Type>,1,1>;
  link->d_ScatterAndMaxloc = ScatterAndOp<Type,Maxloc<Type>,1,1>;
  link->d_ScatterAndMinloc = ScatterAndOp<Type,Minloc<Type>,1,1>;
  /* Atomics for pair types are not implemented yet */
}

template<typename Type,PetscInt BS,PetscInt EQ>
static void PackInit_DumbType(PetscSFLink link)
{
  link->d_Pack             = Pack<Type,BS,EQ>;
  link->d_UnpackAndInsert  = UnpackAndOp<Type,Insert<Type>,BS,EQ>;
  link->d_ScatterAndInsert = ScatterAndInsert<Type,BS,EQ>;
  /* Atomics for dumb types are not implemented yet */
}

/*====================================================================================*/
/*                Main driver to init MPI datatype on device                          */
/*====================================================================================*/

/* Some fields of link are initialized by PetscSFPackSetUp_Host. This routine only does what needed on device */
PetscErrorCode PetscSFLinkSetUp_Device(PetscSF sf,PetscSFLink link,MPI_Datatype unit)
{
  PetscErrorCode ierr;
  hipError_t    err;
  PetscInt       nSignedChar=0,nUnsignedChar=0,nInt=0,nPetscInt=0,nPetscReal=0;
  PetscBool      is2Int,is2PetscInt;
#if defined(PETSC_HAVE_COMPLEX)
  PetscInt       nPetscComplex=0;
#endif

  PetscFunctionBegin;
  if (link->deviceinited) PetscFunctionReturn(0);
  ierr = MPIPetsc_Type_compare_contig(unit,MPI_SIGNED_CHAR,  &nSignedChar);CHKERRQ(ierr);
  ierr = MPIPetsc_Type_compare_contig(unit,MPI_UNSIGNED_CHAR,&nUnsignedChar);CHKERRQ(ierr);
  /* MPI_CHAR is treated below as a dumb type that does not support reduction according to MPI standard */
  ierr = MPIPetsc_Type_compare_contig(unit,MPI_INT,  &nInt);CHKERRQ(ierr);
  ierr = MPIPetsc_Type_compare_contig(unit,MPIU_INT, &nPetscInt);CHKERRQ(ierr);
  ierr = MPIPetsc_Type_compare_contig(unit,MPIU_REAL,&nPetscReal);CHKERRQ(ierr);
#if defined(PETSC_HAVE_COMPLEX)
  ierr = MPIPetsc_Type_compare_contig(unit,MPIU_COMPLEX,&nPetscComplex);CHKERRQ(ierr);
#endif
  ierr = MPIPetsc_Type_compare(unit,MPI_2INT,&is2Int);CHKERRQ(ierr);
  ierr = MPIPetsc_Type_compare(unit,MPIU_2INT,&is2PetscInt);CHKERRQ(ierr);

  if (is2Int) {
    PackInit_PairType<PairInt>(link);
  } else if (is2PetscInt) { /* TODO: when is2PetscInt and nPetscInt=2, we don't know which path to take. The two paths support different ops. */
    PackInit_PairType<PairPetscInt>(link);
  } else if (nPetscReal) {
    if      (nPetscReal == 8) PackInit_RealType<PetscReal,8,1>(link); else if (nPetscReal%8 == 0) PackInit_RealType<PetscReal,8,0>(link);
    else if (nPetscReal == 4) PackInit_RealType<PetscReal,4,1>(link); else if (nPetscReal%4 == 0) PackInit_RealType<PetscReal,4,0>(link);
    else if (nPetscReal == 2) PackInit_RealType<PetscReal,2,1>(link); else if (nPetscReal%2 == 0) PackInit_RealType<PetscReal,2,0>(link);
    else if (nPetscReal == 1) PackInit_RealType<PetscReal,1,1>(link); else if (nPetscReal%1 == 0) PackInit_RealType<PetscReal,1,0>(link);
  } else if (nPetscInt) {
    if      (nPetscInt == 8) PackInit_IntegerType<PetscInt,8,1>(link); else if (nPetscInt%8 == 0) PackInit_IntegerType<PetscInt,8,0>(link);
    else if (nPetscInt == 4) PackInit_IntegerType<PetscInt,4,1>(link); else if (nPetscInt%4 == 0) PackInit_IntegerType<PetscInt,4,0>(link);
    else if (nPetscInt == 2) PackInit_IntegerType<PetscInt,2,1>(link); else if (nPetscInt%2 == 0) PackInit_IntegerType<PetscInt,2,0>(link);
    else if (nPetscInt == 1) PackInit_IntegerType<PetscInt,1,1>(link); else if (nPetscInt%1 == 0) PackInit_IntegerType<PetscInt,1,0>(link);
#if defined(PETSC_USE_64BIT_INDICES)
  } else if (nInt) {
    if      (nInt == 8) PackInit_IntegerType<int,8,1>(link); else if (nInt%8 == 0) PackInit_IntegerType<int,8,0>(link);
    else if (nInt == 4) PackInit_IntegerType<int,4,1>(link); else if (nInt%4 == 0) PackInit_IntegerType<int,4,0>(link);
    else if (nInt == 2) PackInit_IntegerType<int,2,1>(link); else if (nInt%2 == 0) PackInit_IntegerType<int,2,0>(link);
    else if (nInt == 1) PackInit_IntegerType<int,1,1>(link); else if (nInt%1 == 0) PackInit_IntegerType<int,1,0>(link);
#endif
  } else if (nSignedChar) {
    if      (nSignedChar == 8) PackInit_IntegerType<SignedChar,8,1>(link); else if (nSignedChar%8 == 0) PackInit_IntegerType<SignedChar,8,0>(link);
    else if (nSignedChar == 4) PackInit_IntegerType<SignedChar,4,1>(link); else if (nSignedChar%4 == 0) PackInit_IntegerType<SignedChar,4,0>(link);
    else if (nSignedChar == 2) PackInit_IntegerType<SignedChar,2,1>(link); else if (nSignedChar%2 == 0) PackInit_IntegerType<SignedChar,2,0>(link);
    else if (nSignedChar == 1) PackInit_IntegerType<SignedChar,1,1>(link); else if (nSignedChar%1 == 0) PackInit_IntegerType<SignedChar,1,0>(link);
  }  else if (nUnsignedChar) {
    if      (nUnsignedChar == 8) PackInit_IntegerType<UnsignedChar,8,1>(link); else if (nUnsignedChar%8 == 0) PackInit_IntegerType<UnsignedChar,8,0>(link);
    else if (nUnsignedChar == 4) PackInit_IntegerType<UnsignedChar,4,1>(link); else if (nUnsignedChar%4 == 0) PackInit_IntegerType<UnsignedChar,4,0>(link);
    else if (nUnsignedChar == 2) PackInit_IntegerType<UnsignedChar,2,1>(link); else if (nUnsignedChar%2 == 0) PackInit_IntegerType<UnsignedChar,2,0>(link);
    else if (nUnsignedChar == 1) PackInit_IntegerType<UnsignedChar,1,1>(link); else if (nUnsignedChar%1 == 0) PackInit_IntegerType<UnsignedChar,1,0>(link);
#if defined(PETSC_HAVE_COMPLEX)
  } else if (nPetscComplex) {
    if      (nPetscComplex == 8) PackInit_ComplexType<PetscComplex,8,1>(link); else if (nPetscComplex%8 == 0) PackInit_ComplexType<PetscComplex,8,0>(link);
    else if (nPetscComplex == 4) PackInit_ComplexType<PetscComplex,4,1>(link); else if (nPetscComplex%4 == 0) PackInit_ComplexType<PetscComplex,4,0>(link);
    else if (nPetscComplex == 2) PackInit_ComplexType<PetscComplex,2,1>(link); else if (nPetscComplex%2 == 0) PackInit_ComplexType<PetscComplex,2,0>(link);
    else if (nPetscComplex == 1) PackInit_ComplexType<PetscComplex,1,1>(link); else if (nPetscComplex%1 == 0) PackInit_ComplexType<PetscComplex,1,0>(link);
#endif
  } else {
    MPI_Aint lb,nbyte;
    ierr = MPI_Type_get_extent(unit,&lb,&nbyte);CHKERRQ(ierr);
    if (lb != 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"Datatype with nonzero lower bound %ld\n",(long)lb);
    if (nbyte % sizeof(int)) { /* If the type size is not multiple of int */
      if      (nbyte == 4) PackInit_DumbType<char,4,1>(link); else if (nbyte%4 == 0) PackInit_DumbType<char,4,0>(link);
      else if (nbyte == 2) PackInit_DumbType<char,2,1>(link); else if (nbyte%2 == 0) PackInit_DumbType<char,2,0>(link);
      else if (nbyte == 1) PackInit_DumbType<char,1,1>(link); else if (nbyte%1 == 0) PackInit_DumbType<char,1,0>(link);
    } else {
      nInt = nbyte / sizeof(int);
      if      (nInt == 8) PackInit_DumbType<int,8,1>(link); else if (nInt%8 == 0) PackInit_DumbType<int,8,0>(link);
      else if (nInt == 4) PackInit_DumbType<int,4,1>(link); else if (nInt%4 == 0) PackInit_DumbType<int,4,0>(link);
      else if (nInt == 2) PackInit_DumbType<int,2,1>(link); else if (nInt%2 == 0) PackInit_DumbType<int,2,0>(link);
      else if (nInt == 1) PackInit_DumbType<int,1,1>(link); else if (nInt%1 == 0) PackInit_DumbType<int,1,0>(link);
    }
  }

  if (!sf->use_default_stream) {err = hipStreamCreate(&link->stream);CHKERRCUDA(err);}
  if (!sf->maxResidentThreadsPerGPU) { /* Not initialized */
    int                   device;
    struct hipDeviceProp_t props;
    err = hipGetDevice(&device);CHKERRCUDA(err);
    err = hipGetDeviceProperties(&props,device);CHKERRCUDA(err);
    sf->maxResidentThreadsPerGPU = props.maxThreadsPerMultiProcessor*props.multiProcessorCount;
  }
  link->maxResidentThreadsPerGPU = sf->maxResidentThreadsPerGPU;
  link->deviceinited             = PETSC_TRUE;
  PetscFunctionReturn(0);
}
