/*
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
   SLEPc - Scalable Library for Eigenvalue Problem Computations
   Copyright (c) 2002-2020, Universitat Politecnica de Valencia, Spain

   This file is part of SLEPc.
   SLEPc is distributed under a 2-clause BSD license (see LICENSE).
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
*/
/*
   BV implemented as a single Vec (CUDA version)
*/

#include <slepc/private/bvimpl.h>
#include "../src/sys/classes/bv/impls/svec/svec.h"
#include <petsccublas.h>

#if defined(PETSC_USE_COMPLEX)
#include <thrust/device_ptr.h>
#endif

#define BLOCKSIZE 64

/*
    B := alpha*A + beta*B

    A,B are nxk (ld=n)
 */
static PetscErrorCode BVAXPY_BLAS_CUDA(BV bv,PetscInt n_,PetscInt k_,PetscScalar alpha,const PetscScalar *d_A,PetscScalar beta,PetscScalar *d_B)
{
  PetscErrorCode ierr;
  PetscBLASInt   m,one=1;
  hipblasStatus_t cberr;
  hipblasHandle_t cublasv2handle;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(n_*k_,&m);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (beta!=(PetscScalar)1.0) {
    cberr = cublasXscal(cublasv2handle,m,&beta,d_B,one);CHKERRCUBLAS(cberr);
    ierr = PetscLogGpuFlops(1.0*m);CHKERRQ(ierr);
  }
  cberr = cublasXaxpy(cublasv2handle,m,&alpha,d_A,one,d_B,one);CHKERRCUBLAS(cberr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*m);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*
    C := alpha*A*B + beta*C
*/
PetscErrorCode BVMult_Svec_CUDA(BV Y,PetscScalar alpha,PetscScalar beta,BV X,Mat Q)
{
  PetscErrorCode    ierr;
  BV_SVEC           *y = (BV_SVEC*)Y->data,*x = (BV_SVEC*)X->data;
  const PetscScalar *d_px,*d_A;
  PetscScalar       *d_py,*q,*d_q,*d_B,*d_C;
  PetscInt          ldq,mq;
  PetscBLASInt      m,n,k,ldq_;
  hipblasStatus_t    cberr;
  hipError_t       cerr;
  hipblasHandle_t    cublasv2handle;

  PetscFunctionBegin;
  if (!Y->n) PetscFunctionReturn(0);
  ierr = VecCUDAGetArrayRead(x->v,&d_px);CHKERRQ(ierr);
  if (beta==(PetscScalar)0.0) {
    ierr = VecCUDAGetArrayWrite(y->v,&d_py);CHKERRQ(ierr);
  } else {
    ierr = VecCUDAGetArray(y->v,&d_py);CHKERRQ(ierr);
  }
  d_A = d_px+(X->nc+X->l)*X->n;
  d_C = d_py+(Y->nc+Y->l)*Y->n;
  if (Q) {
    ierr = PetscBLASIntCast(Y->n,&m);CHKERRQ(ierr);
    ierr = PetscBLASIntCast(Y->k-Y->l,&n);CHKERRQ(ierr);
    ierr = PetscBLASIntCast(X->k-X->l,&k);CHKERRQ(ierr);
    ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
    ierr = MatGetSize(Q,&ldq,&mq);CHKERRQ(ierr);
    ierr = PetscBLASIntCast(ldq,&ldq_);CHKERRQ(ierr);
    ierr = MatDenseGetArray(Q,&q);CHKERRQ(ierr);
    cerr = hipMalloc((void**)&d_q,ldq*mq*sizeof(PetscScalar));CHKERRCUDA(cerr);
    cerr = hipMemcpy(d_q,q,ldq*mq*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    ierr = PetscLogCpuToGpu(ldq*mq*sizeof(PetscScalar));CHKERRQ(ierr);
    d_B = d_q+Y->l*ldq+X->l;
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&alpha,d_A,m,d_B,ldq_,&beta,d_C,m);CHKERRCUBLAS(cberr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = MatDenseRestoreArray(Q,&q);CHKERRQ(ierr);
    cerr = hipFree(d_q);CHKERRCUDA(cerr);
    ierr = PetscLogGpuFlops(2.0*m*n*k);CHKERRQ(ierr);
  } else {
    ierr = BVAXPY_BLAS_CUDA(Y,Y->n,Y->k-Y->l,alpha,d_A,beta,d_C);CHKERRQ(ierr);
  }
  ierr = VecCUDARestoreArrayRead(x->v,&d_px);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(y->v,&d_py);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*
    y := alpha*A*x + beta*y
*/
PetscErrorCode BVMultVec_Svec_CUDA(BV X,PetscScalar alpha,PetscScalar beta,Vec y,PetscScalar *q)
{
  PetscErrorCode    ierr;
  BV_SVEC           *x = (BV_SVEC*)X->data;
  const PetscScalar *d_px,*d_A;
  PetscScalar       *d_py,*d_q,*d_x,*d_y;
  PetscBLASInt      n,k,one=1;
  hipblasStatus_t    cberr;
  hipblasHandle_t    cublasv2handle;
  hipError_t       cerr;

  PetscFunctionBegin;
  ierr = PetscBLASIntCast(X->n,&n);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(X->k-X->l,&k);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(x->v,&d_px);CHKERRQ(ierr);
  if (beta==(PetscScalar)0.0) {
    ierr = VecCUDAGetArrayWrite(y,&d_py);CHKERRQ(ierr);
  } else {
    ierr = VecCUDAGetArray(y,&d_py);CHKERRQ(ierr);
  }
  if (!q) {
    ierr = VecCUDAGetArray(X->buffer,&d_q);CHKERRQ(ierr);
  } else {
    cerr = hipMalloc((void**)&d_q,k*sizeof(PetscScalar));CHKERRCUDA(cerr);
    cerr = hipMemcpy(d_q,q,k*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    ierr = PetscLogCpuToGpu(k*sizeof(PetscScalar));CHKERRQ(ierr);
  }
  d_A = d_px+(X->nc+X->l)*X->n;
  d_x = d_q;
  d_y = d_py;
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  cberr = cublasXgemv(cublasv2handle,HIPBLAS_OP_N,n,k,&alpha,d_A,n,d_x,one,&beta,d_y,one);CHKERRCUBLAS(cberr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(x->v,&d_px);CHKERRQ(ierr);
  if (beta==(PetscScalar)0.0) {
    ierr = VecCUDARestoreArrayWrite(y,&d_py);CHKERRQ(ierr);
  } else {
    ierr = VecCUDARestoreArray(y,&d_py);CHKERRQ(ierr);
  }
  if (!q) {
    ierr = VecCUDARestoreArray(X->buffer,&d_q);CHKERRQ(ierr);
  } else {
    cerr = hipFree(d_q);CHKERRCUDA(cerr);
  }
  ierr = PetscLogGpuFlops(2.0*n*k);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*
    A(:,s:e-1) := A*B(:,s:e-1)
*/
PetscErrorCode BVMultInPlace_Svec_CUDA(BV V,Mat Q,PetscInt s,PetscInt e)
{
  PetscErrorCode ierr;
  BV_SVEC        *ctx = (BV_SVEC*)V->data;
  PetscScalar    *d_pv,*q,*d_q,*d_A,*d_B,*d_work,sone=1.0,szero=0.0;
  PetscInt       j,ldq,nq;
  PetscBLASInt   m,n,k,l,ldq_,bs=BLOCKSIZE;
  hipblasStatus_t cberr;
  size_t         freemem,totmem;
  hipblasHandle_t cublasv2handle;
  hipError_t    cerr;

  PetscFunctionBegin;
  if (!V->n) PetscFunctionReturn(0);
  ierr = PetscBLASIntCast(V->n,&m);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(e-s,&n);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(V->k-V->l,&k);CHKERRQ(ierr);
  ierr = MatGetSize(Q,&ldq,&nq);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(ldq,&ldq_);CHKERRQ(ierr);
  ierr = VecCUDAGetArray(ctx->v,&d_pv);CHKERRQ(ierr);
  ierr = MatDenseGetArray(Q,&q);CHKERRQ(ierr);
  cerr = hipMalloc((void**)&d_q,ldq*nq*sizeof(PetscScalar));CHKERRCUDA(cerr);
  cerr = hipMemcpy(d_q,q,ldq*nq*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
  ierr = PetscLogCpuToGpu(ldq*nq*sizeof(PetscScalar));CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  /* try to allocate the whole matrix */
  cerr = hipMemGetInfo(&freemem,&totmem);CHKERRCUDA(cerr);
  if (freemem>=m*n*sizeof(PetscScalar)) {
    cerr = hipMalloc((void**)&d_work,m*n*sizeof(PetscScalar));CHKERRCUDA(cerr);
    d_A = d_pv+(V->nc+V->l)*m;
    d_B = d_q+V->l*ldq+V->l+(s-V->l)*ldq;
    cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&sone,d_A,m,d_B,ldq_,&szero,d_work,m);CHKERRCUBLAS(cberr);
    for (j=0;j<n;j++) {
      cerr = hipMemcpy(d_A+(s-V->l+j)*m,d_work+(j*m),m*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
    }
  } else {
    bs = freemem/(m*sizeof(PetscScalar));
    cerr = hipMalloc((void**)&d_work,bs*n*sizeof(PetscScalar));CHKERRCUDA(cerr);
    l = m % bs;
    if (l) {
      d_A = d_pv+(V->nc+V->l)*m;
      d_B = d_q+V->l*ldq+V->l+(s-V->l)*ldq;
      cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_N,HIPBLAS_OP_N,l,n,k,&sone,d_A,m,d_B,ldq_,&szero,d_work,l);CHKERRCUBLAS(cberr);
      for (j=0;j<n;j++) {
        cerr = hipMemcpy(d_A+(s-V->l+j)*m,d_work+(j*l),l*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
      }
    }
    for (;l<m;l+=bs) {
      d_A = d_pv+(V->nc+V->l)*m+l;
      d_B = d_q+V->l*ldq+V->l+(s-V->l)*ldq;
      cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_N,HIPBLAS_OP_N,bs,n,k,&sone,d_A,m,d_B,ldq_,&szero,d_work,bs);CHKERRCUBLAS(cberr);
      for (j=0;j<n;j++) {
        cerr = hipMemcpy(d_A+(s-V->l+j)*m,d_work+(j*bs),bs*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
      }
    }
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  cerr = WaitForGPU();CHKERRCUDA(cerr);
  ierr = MatDenseRestoreArray(Q,&q);CHKERRQ(ierr);
  cerr = hipFree(d_q);CHKERRCUDA(cerr);
  cerr = hipFree(d_work);CHKERRCUDA(cerr);
  ierr = VecCUDARestoreArray(ctx->v,&d_pv);CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*m*n*k);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*
    A(:,s:e-1) := A*B(:,s:e-1)
*/
PetscErrorCode BVMultInPlaceTranspose_Svec_CUDA(BV V,Mat Q,PetscInt s,PetscInt e)
{
  PetscErrorCode ierr;
  BV_SVEC        *ctx = (BV_SVEC*)V->data;
  PetscScalar    *d_pv,*q,*d_q,*d_A,*d_B,*d_work,sone=1.0,szero=0.0;
  PetscInt       j,ldq,nq;
  PetscBLASInt   m,n,k,ldq_;
  hipblasStatus_t cberr;
  hipblasHandle_t cublasv2handle;
  hipError_t    cerr;

  PetscFunctionBegin;
  if (!V->n) PetscFunctionReturn(0);
  ierr = PetscBLASIntCast(V->n,&m);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(e-s,&n);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(V->k-V->l,&k);CHKERRQ(ierr);
  ierr = MatGetSize(Q,&ldq,&nq);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(ldq,&ldq_);CHKERRQ(ierr);
  ierr = VecCUDAGetArray(ctx->v,&d_pv);CHKERRQ(ierr);
  ierr = MatDenseGetArray(Q,&q);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  cerr = hipMalloc((void**)&d_q,ldq*nq*sizeof(PetscScalar));CHKERRCUDA(cerr);
  cerr = hipMemcpy(d_q,q,ldq*nq*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
  ierr = PetscLogCpuToGpu(ldq*nq*sizeof(PetscScalar));CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  cerr = hipMalloc((void**)&d_work,m*n*sizeof(PetscScalar));CHKERRCUDA(cerr);
  d_A = d_pv+(V->nc+V->l)*m;
  d_B = d_q+V->l*ldq+s;
  cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_N,HIPBLAS_OP_C,m,n,k,&sone,d_A,m,d_B,ldq_,&szero,d_work,m);CHKERRCUBLAS(cberr);
  for (j=0;j<n;j++) {
    cerr = hipMemcpy(d_A+(s-V->l+j)*m,d_work+(j*m),m*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  cerr = WaitForGPU();CHKERRCUDA(cerr);
  ierr = MatDenseRestoreArray(Q,&q);CHKERRQ(ierr);
  cerr = hipFree(d_q);CHKERRCUDA(cerr);
  cerr = hipFree(d_work);CHKERRCUDA(cerr);
  ierr = VecCUDARestoreArray(ctx->v,&d_pv);CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*m*n*k);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*
    C := A'*B
*/
PetscErrorCode BVDot_Svec_CUDA(BV X,BV Y,Mat M)
{
  PetscErrorCode    ierr;
  BV_SVEC           *x = (BV_SVEC*)X->data,*y = (BV_SVEC*)Y->data;
  const PetscScalar *d_px,*d_py,*d_A,*d_B;
  PetscScalar       *pm,*d_work,sone=1.0,szero=0.0,*C,*CC;
  PetscInt          j,ldm;
  PetscBLASInt      m,n,k,ldm_;
  PetscMPIInt       len;
  hipblasStatus_t    cberr;
  hipblasHandle_t    cublasv2handle;
  hipError_t       cerr;

  PetscFunctionBegin;
  ierr = PetscBLASIntCast(Y->k-Y->l,&m);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(X->k-X->l,&n);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(X->n,&k);CHKERRQ(ierr);
  ierr = MatGetSize(M,&ldm,NULL);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(ldm,&ldm_);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(x->v,&d_px);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(y->v,&d_py);CHKERRQ(ierr);
  ierr = MatDenseGetArray(M,&pm);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  cerr = hipMalloc((void**)&d_work,m*n*sizeof(PetscScalar));CHKERRCUDA(cerr);
  d_A = d_py+(Y->nc+Y->l)*Y->n;
  d_B = d_px+(X->nc+X->l)*X->n;
  C = pm+X->l*ldm+Y->l;
  if (x->mpi) {
    if (ldm==m) {
      ierr = BVAllocateWork_Private(X,m*n);CHKERRQ(ierr);
      if (k) {
        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
        cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_C,HIPBLAS_OP_N,m,n,k,&sone,d_A,k,d_B,k,&szero,d_work,ldm_);CHKERRCUBLAS(cberr);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
        cerr = hipMemcpy(X->work,d_work,m*n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
        ierr = PetscLogGpuToCpu(m*n*sizeof(PetscScalar));CHKERRQ(ierr);
      } else {
        ierr = PetscArrayzero(X->work,m*n);CHKERRQ(ierr);
      }
      ierr = PetscMPIIntCast(m*n,&len);CHKERRQ(ierr);
      ierr = MPI_Allreduce(X->work,C,len,MPIU_SCALAR,MPIU_SUM,PetscObjectComm((PetscObject)X));CHKERRQ(ierr);
    } else {
      ierr = BVAllocateWork_Private(X,2*m*n);CHKERRQ(ierr);
      CC = X->work+m*n;
      if (k) {
        ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
        cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_C,HIPBLAS_OP_N,m,n,k,&sone,d_A,k,d_B,k,&szero,d_work,m);CHKERRCUBLAS(cberr);
        ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
        cerr = hipMemcpy(X->work,d_work,m*n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
        ierr = PetscLogGpuToCpu(m*n*sizeof(PetscScalar));CHKERRQ(ierr);
      } else {
        ierr = PetscArrayzero(X->work,m*n);CHKERRQ(ierr);
      }
      ierr = PetscMPIIntCast(m*n,&len);CHKERRQ(ierr);
      ierr = MPI_Allreduce(X->work,CC,len,MPIU_SCALAR,MPIU_SUM,PetscObjectComm((PetscObject)X));CHKERRQ(ierr);
      for (j=0;j<n;j++) {
        ierr = PetscArraycpy(C+j*ldm,CC+j*m,m);CHKERRQ(ierr);
      }
    }
  } else {
    if (k) {
      ierr = BVAllocateWork_Private(X,m*n);CHKERRQ(ierr);
      ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
      cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_C,HIPBLAS_OP_N,m,n,k,&sone,d_A,k,d_B,k,&szero,d_work,m);CHKERRCUBLAS(cberr);
      ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
      cerr = hipMemcpy(X->work,d_work,m*n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
      ierr = PetscLogGpuToCpu(m*n*sizeof(PetscScalar));CHKERRQ(ierr);
      for (j=0;j<n;j++) {
        ierr = PetscArraycpy(C+j*ldm,X->work+j*m,m);CHKERRQ(ierr);
      }
    }
  }
  cerr = WaitForGPU();CHKERRCUDA(cerr);
  cerr = hipFree(d_work);CHKERRCUDA(cerr);
  ierr = MatDenseRestoreArray(M,&pm);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(x->v,&d_px);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(y->v,&d_py);CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*m*n*k);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#if defined(PETSC_USE_COMPLEX)
struct conjugate
{
  __host__ __device__
    PetscScalar operator()(PetscScalar x)
    {
      return PetscConj(x);
    }
};

PetscErrorCode ConjugateCudaArray(PetscScalar *a, PetscInt n)
{
  hipError_t                     cerr;
  thrust::device_ptr<PetscScalar> ptr;

  PetscFunctionBegin;
  try {
    ptr = thrust::device_pointer_cast(a);
    thrust::transform(ptr,ptr+n,ptr,conjugate());
    cerr = WaitForGPU();CHKERRCUDA(cerr);
  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Thrust error: %s", ex);
  }
  PetscFunctionReturn(0);
}
#endif

/*
    y := A'*x computed as y' := x'*A
*/
PetscErrorCode BVDotVec_Svec_CUDA(BV X,Vec y,PetscScalar *q)
{
  PetscErrorCode    ierr;
  BV_SVEC           *x = (BV_SVEC*)X->data;
  const PetscScalar *d_A,*d_x,*d_px,*d_py;
  PetscScalar       *d_work,szero=0.0,sone=1.0,*qq=q;
  PetscBLASInt      n,k,one=1;
  PetscMPIInt       len;
  Vec               z = y;
  hipblasStatus_t    cberr;
  hipblasHandle_t    cublasv2handle;
  hipError_t       cerr;

  PetscFunctionBegin;
  ierr = PetscBLASIntCast(X->n,&n);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(X->k-X->l,&k);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  if (X->matrix) {
    ierr = BV_IPMatMult(X,y);CHKERRQ(ierr);
    z = X->Bx;
  }
  ierr = VecCUDAGetArrayRead(x->v,&d_px);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(z,&d_py);CHKERRQ(ierr);
  if (!q) {
    ierr = VecCUDAGetArrayWrite(X->buffer,&d_work);CHKERRQ(ierr);
  } else {
    cerr = hipMalloc((void**)&d_work,k*sizeof(PetscScalar));CHKERRCUDA(cerr);
  }
  d_A = d_px+(X->nc+X->l)*X->n;
  d_x = d_py;
  if (x->mpi) {
    ierr = BVAllocateWork_Private(X,k);CHKERRQ(ierr);
    if (n) {
      ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
      cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_C,HIPBLAS_OP_N,one,k,n,&sone,d_x,n,d_A,n,&szero,d_work,one);CHKERRCUBLAS(cberr);
      ierr = ConjugateCudaArray(d_work,k);CHKERRQ(ierr);
#else
      cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_N,HIPBLAS_OP_N,one,k,n,&sone,d_x,one,d_A,n,&szero,d_work,one);CHKERRCUBLAS(cberr);
#endif
      ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
      cerr = hipMemcpy(X->work,d_work,k*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
      ierr = PetscLogGpuToCpu(k*sizeof(PetscScalar));CHKERRQ(ierr);
    } else {
      ierr = PetscArrayzero(X->work,k);CHKERRQ(ierr);
    }
    if (!q) {
      ierr = VecCUDARestoreArrayWrite(X->buffer,&d_work);CHKERRQ(ierr);
      ierr = VecGetArray(X->buffer,&qq);CHKERRQ(ierr);
    } else {
      cerr = hipFree(d_work);CHKERRCUDA(cerr);
    }
    ierr = PetscMPIIntCast(k,&len);CHKERRQ(ierr);
    ierr = MPI_Allreduce(X->work,qq,len,MPIU_SCALAR,MPIU_SUM,PetscObjectComm((PetscObject)X));CHKERRQ(ierr);
    if (!q) { ierr = VecRestoreArray(X->buffer,&qq);CHKERRQ(ierr); }
  } else {
    if (n) {
      ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
      cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_C,HIPBLAS_OP_N,one,k,n,&sone,d_x,n,d_A,n,&szero,d_work,one);CHKERRCUBLAS(cberr);
      ierr = ConjugateCudaArray(d_work,k);CHKERRQ(ierr);
#else
      cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_N,HIPBLAS_OP_N,one,k,n,&sone,d_x,one,d_A,n,&szero,d_work,one);CHKERRCUBLAS(cberr);
#endif
      ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    }
    if (!q) {
      ierr = VecCUDARestoreArrayWrite(X->buffer,&d_work);CHKERRQ(ierr);
    } else {
      cerr = hipMemcpy(q,d_work,k*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
      ierr = PetscLogGpuToCpu(k*sizeof(PetscScalar));CHKERRQ(ierr);
      cerr = hipFree(d_work);CHKERRCUDA(cerr);
    }
  }
  cerr = WaitForGPU();CHKERRCUDA(cerr);
  ierr = VecCUDARestoreArrayRead(z,&d_py);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(x->v,&d_px);CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*n*k);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*
    y := A'*x computed as y' := x'*A
*/
PetscErrorCode BVDotVec_Local_Svec_CUDA(BV X,Vec y,PetscScalar *m)
{
  PetscErrorCode    ierr;
  BV_SVEC           *x = (BV_SVEC*)X->data;
  const PetscScalar *d_A,*d_x,*d_px,*d_py;
  PetscScalar       *d_y,szero=0.0,sone=1.0;
  PetscBLASInt      n,k,one=1;
  Vec               z = y;
  hipblasStatus_t    cberr;
  hipblasHandle_t    cublasv2handle;
  hipError_t       cerr;

  PetscFunctionBegin;
  ierr = PetscBLASIntCast(X->n,&n);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(X->k-X->l,&k);CHKERRQ(ierr);
  if (X->matrix) {
    ierr = BV_IPMatMult(X,y);CHKERRQ(ierr);
    z = X->Bx;
  }
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(x->v,&d_px);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(z,&d_py);CHKERRQ(ierr);
  d_A = d_px+(X->nc+X->l)*X->n;
  d_x = d_py;
  if (n) {
    cerr = hipMalloc((void**)&d_y,k*sizeof(PetscScalar));CHKERRCUDA(cerr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
    cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_C,HIPBLAS_OP_N,one,k,n,&sone,d_x,n,d_A,n,&szero,d_y,one);CHKERRCUBLAS(cberr);
    ierr = ConjugateCudaArray(d_y,k);CHKERRQ(ierr);
#else
    cberr = cublasXgemm(cublasv2handle,HIPBLAS_OP_N,HIPBLAS_OP_N,one,k,n,&sone,d_x,one,d_A,n,&szero,d_y,one);CHKERRCUBLAS(cberr);
#endif
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    cerr = hipMemcpy(m,d_y,k*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    ierr = PetscLogGpuToCpu(k*sizeof(PetscScalar));CHKERRQ(ierr);
    cerr = hipFree(d_y);CHKERRCUDA(cerr);
  }
  ierr = VecCUDARestoreArrayRead(z,&d_py);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(x->v,&d_px);CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*n*k);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*
    Scale n scalars
*/
PetscErrorCode BVScale_Svec_CUDA(BV bv,PetscInt j,PetscScalar alpha)
{
  PetscErrorCode ierr;
  BV_SVEC        *ctx = (BV_SVEC*)bv->data;
  PetscScalar    *d_array, *d_A;
  PetscBLASInt   n,one=1;
  hipblasStatus_t cberr;
  hipblasHandle_t cublasv2handle;
  hipError_t    cerr;

  PetscFunctionBegin;
  ierr = VecCUDAGetArray(ctx->v,&d_array);CHKERRQ(ierr);
  if (j<0) {
    d_A = d_array+(bv->nc+bv->l)*bv->n;
    ierr = PetscBLASIntCast((bv->k-bv->l)*bv->n,&n);CHKERRQ(ierr);
  } else {
    d_A = d_array+(bv->nc+j)*bv->n;
    ierr = PetscBLASIntCast(bv->n,&n);CHKERRQ(ierr);
  }
  if (alpha == (PetscScalar)0.0) {
    cerr = hipMemset(d_A,0,n*sizeof(PetscScalar));CHKERRCUDA(cerr);
  } else if (alpha != (PetscScalar)1.0) {
    ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXscal(cublasv2handle,n,&alpha,d_A,one);CHKERRCUBLAS(cberr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(1.0*n);CHKERRQ(ierr);
  }
  ierr = VecCUDARestoreArray(ctx->v,&d_array);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode BVMatMult_Svec_CUDA(BV V,Mat A,BV W)
{
  PetscErrorCode    ierr;
  BV_SVEC           *v = (BV_SVEC*)V->data,*w = (BV_SVEC*)W->data;
  const PetscScalar *d_pv;
  PetscScalar       *d_pw;
  PetscInt          j;

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayRead(v->v,&d_pv);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayWrite(w->v,&d_pw);CHKERRQ(ierr);
  for (j=0;j<V->k-V->l;j++) {
    ierr = VecCUDAPlaceArray(V->cv[1],(PetscScalar *)d_pv+(V->nc+V->l+j)*V->n);CHKERRQ(ierr);
    ierr = VecCUDAPlaceArray(W->cv[1],d_pw+(W->nc+W->l+j)*W->n);CHKERRQ(ierr);
    ierr = MatMult(A,V->cv[1],W->cv[1]);CHKERRQ(ierr);
    ierr = VecCUDAResetArray(V->cv[1]);CHKERRQ(ierr);
    ierr = VecCUDAResetArray(W->cv[1]);CHKERRQ(ierr);
  }
  ierr = VecCUDARestoreArrayRead(v->v,&d_pv);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(w->v,&d_pw);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode BVCopy_Svec_CUDA(BV V,BV W)
{
  PetscErrorCode    ierr;
  BV_SVEC           *v = (BV_SVEC*)V->data,*w = (BV_SVEC*)W->data;
  const PetscScalar *d_pv,*d_pvc;
  PetscScalar       *d_pw,*d_pwc;
  hipError_t       cerr;

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayRead(v->v,&d_pv);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayWrite(w->v,&d_pw);CHKERRQ(ierr);
  d_pvc = d_pv+(V->nc+V->l)*V->n;
  d_pwc = d_pw+(W->nc+W->l)*W->n;
  cerr = hipMemcpy(d_pwc,d_pvc,(V->k-V->l)*V->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
  ierr = VecCUDARestoreArrayRead(v->v,&d_pv);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(w->v,&d_pw);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode BVCopyColumn_Svec_CUDA(BV V,PetscInt j,PetscInt i)
{
  PetscErrorCode ierr;
  BV_SVEC        *v = (BV_SVEC*)V->data;
  PetscScalar    *d_pv;
  hipError_t    cerr;

  PetscFunctionBegin;
  ierr = VecCUDAGetArray(v->v,&d_pv);CHKERRQ(ierr);
  cerr = hipMemcpy(d_pv+(V->nc+i)*V->n,d_pv+(V->nc+j)*V->n,V->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
  ierr = VecCUDARestoreArray(v->v,&d_pv);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode BVResize_Svec_CUDA(BV bv,PetscInt m,PetscBool copy)
{
  PetscErrorCode    ierr;
  BV_SVEC           *ctx = (BV_SVEC*)bv->data;
  const PetscScalar *d_pv;
  PetscScalar       *d_pnew,*d_ptr;
  PetscInt          bs,lsplit;
  Vec               vnew,vpar;
  char              str[50];
  hipError_t       cerr;
  BV                parent;

  PetscFunctionBegin;
  if (bv->issplit==2) {
    parent = bv->splitparent;
    lsplit = parent->lsplit;
    vpar = ((BV_SVEC*)parent->data)->v;
    ierr = VecCUDAResetArray(ctx->v);CHKERRQ(ierr);
    ierr = VecCUDAGetArray(vpar,&d_ptr);CHKERRQ(ierr);
    ierr = VecCUDAPlaceArray(ctx->v,d_ptr+lsplit*bv->n);CHKERRQ(ierr);
    ierr = VecCUDARestoreArray(vpar,&d_ptr);CHKERRQ(ierr);
  } else if (!bv->issplit) {
    ierr = VecGetBlockSize(bv->t,&bs);CHKERRQ(ierr);
    ierr = VecCreate(PetscObjectComm((PetscObject)bv->t),&vnew);CHKERRQ(ierr);
    ierr = VecSetType(vnew,((PetscObject)bv->t)->type_name);CHKERRQ(ierr);
    ierr = VecSetSizes(vnew,m*bv->n,PETSC_DECIDE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(vnew,bs);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)bv,(PetscObject)vnew);CHKERRQ(ierr);
    if (((PetscObject)bv)->name) {
      ierr = PetscSNPrintf(str,50,"%s_0",((PetscObject)bv)->name);CHKERRQ(ierr);
      ierr = PetscObjectSetName((PetscObject)vnew,str);CHKERRQ(ierr);
    }
    if (copy) {
      ierr = VecCUDAGetArrayRead(ctx->v,&d_pv);CHKERRQ(ierr);
      ierr = VecCUDAGetArrayWrite(vnew,&d_pnew);CHKERRQ(ierr);
      cerr = hipMemcpy(d_pnew,d_pv,PetscMin(m,bv->m)*bv->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
      ierr = VecCUDARestoreArrayRead(ctx->v,&d_pv);CHKERRQ(ierr);
      ierr = VecCUDARestoreArrayWrite(vnew,&d_pnew);CHKERRQ(ierr);
    }
    ierr = VecDestroy(&ctx->v);CHKERRQ(ierr);
    ctx->v = vnew;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode BVGetColumn_Svec_CUDA(BV bv,PetscInt j,Vec *v)
{
  PetscErrorCode ierr;
  BV_SVEC        *ctx = (BV_SVEC*)bv->data;
  PetscScalar    *d_pv;
  PetscInt       l;

  PetscFunctionBegin;
  l = BVAvailableVec;
  ierr = VecCUDAGetArray(ctx->v,&d_pv);CHKERRQ(ierr);
  ierr = VecCUDAPlaceArray(bv->cv[l],d_pv+(bv->nc+j)*bv->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode BVRestoreColumn_Svec_CUDA(BV bv,PetscInt j,Vec *v)
{
  PetscErrorCode ierr;
  BV_SVEC        *ctx = (BV_SVEC*)bv->data;
  PetscInt       l;

  PetscFunctionBegin;
  l = (j==bv->ci[0])? 0: 1;
  ierr = VecCUDAResetArray(bv->cv[l]);CHKERRQ(ierr);
  ierr = VecCUDARestoreArray(ctx->v,NULL);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode BVRestoreSplit_Svec_CUDA(BV bv,BV *L,BV *R)
{
  PetscErrorCode    ierr;
  Vec               v;
  const PetscScalar *d_pv;
  PetscObjectState  lstate,rstate;
  PetscBool         change=PETSC_FALSE;

  PetscFunctionBegin;
  /* force sync flag to PETSC_CUDA_BOTH */
  if (L) {
    ierr = PetscObjectStateGet((PetscObject)*L,&lstate);CHKERRQ(ierr);
    if (lstate != bv->lstate) {
      v = ((BV_SVEC*)bv->L->data)->v;
      ierr = VecCUDAGetArrayRead(v,&d_pv);CHKERRQ(ierr);
      ierr = VecCUDARestoreArrayRead(v,&d_pv);CHKERRQ(ierr);
      change = PETSC_TRUE;
    }
  }
  if (R) {
    ierr = PetscObjectStateGet((PetscObject)*R,&rstate);CHKERRQ(ierr);
    if (rstate != bv->rstate) {
      v = ((BV_SVEC*)bv->R->data)->v;
      ierr = VecCUDAGetArrayRead(v,&d_pv);CHKERRQ(ierr);
      ierr = VecCUDARestoreArrayRead(v,&d_pv);CHKERRQ(ierr);
      change = PETSC_TRUE;
    }
  }
  if (change) {
    v = ((BV_SVEC*)bv->data)->v;
    ierr = VecCUDAGetArray(v,(PetscScalar **)&d_pv);CHKERRQ(ierr);
    ierr = VecCUDARestoreArray(v,(PetscScalar **)&d_pv);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}
