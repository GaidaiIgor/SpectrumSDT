#include "hip/hip_runtime.h"
/*
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
   SLEPc - Scalable Library for Eigenvalue Problem Computations
   Copyright (c) 2002-2020, Universitat Politecnica de Valencia, Spain

   This file is part of SLEPc.
   SLEPc is distributed under a 2-clause BSD license (see LICENSE).
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
*/
/*
   BV orthogonalization routines (CUDA)
*/

#include <slepc/private/bvimpl.h>          /*I   "slepcbv.h"   I*/
#include <slepcblaslapack.h>
#include <petsccublas.h>

/*
   BV_CleanCoefficients_CUDA - Sets to zero all entries of column j of the bv buffer
*/
PetscErrorCode BV_CleanCoefficients_CUDA(BV bv,PetscInt j,PetscScalar *h)
{
  PetscErrorCode ierr;
  PetscScalar    *d_hh,*d_a;
  PetscInt       i;
  hipError_t    cerr;

  PetscFunctionBegin;
  if (!h) {
    ierr = VecCUDAGetArray(bv->buffer,&d_a);CHKERRQ(ierr);
    d_hh = d_a + j*(bv->nc+bv->m);
    cerr = hipMemset(d_hh,0,(bv->nc+j)*sizeof(PetscScalar));CHKERRCUDA(cerr);
    cerr = WaitForGPU();CHKERRCUDA(cerr);
    ierr = VecCUDARestoreArray(bv->buffer,&d_a);CHKERRQ(ierr);
  } else { /* cpu memory */
    for (i=0;i<bv->nc+j;i++) h[i] = 0.0;
  }
  PetscFunctionReturn(0);
}

/*
   BV_AddCoefficients_CUDA - Add the contents of the scratch (0-th column) of the bv buffer
   into column j of the bv buffer
 */
PetscErrorCode BV_AddCoefficients_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscScalar *c)
{
  PetscErrorCode ierr;
  PetscScalar    *d_h,*d_c,sone=1.0;
  PetscInt       i;
  PetscBLASInt   idx,one=1;
  hipblasStatus_t cberr;
  hipblasHandle_t cublasv2handle;

  PetscFunctionBegin;
  if (!h) {
    ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
    ierr = VecCUDAGetArray(bv->buffer,&d_c);CHKERRQ(ierr);
    d_h = d_c + j*(bv->nc+bv->m);
    ierr = PetscBLASIntCast(bv->nc+j,&idx);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXaxpy(cublasv2handle,idx,&sone,d_c,one,d_h,one);CHKERRCUBLAS(cberr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(1.0*bv->nc+j);CHKERRQ(ierr);
    ierr = VecCUDARestoreArray(bv->buffer,&d_c);CHKERRQ(ierr);
  } else { /* cpu memory */
    for (i=0;i<bv->nc+j;i++) h[i] += c[i];
    ierr = PetscLogFlops(1.0*bv->nc+j);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*
   BV_SetValue_CUDA - Sets value in row j (counted after the constraints) of column k
   of the coefficients array
*/
PetscErrorCode BV_SetValue_CUDA(BV bv,PetscInt j,PetscInt k,PetscScalar *h,PetscScalar value)
{
  PetscErrorCode ierr;
  PetscScalar    *d_h,*a;
  hipError_t    cerr;

  PetscFunctionBegin;
  if (!h) {
    ierr = VecCUDAGetArray(bv->buffer,&a);CHKERRQ(ierr);
    d_h = a + k*(bv->nc+bv->m) + bv->nc+j;
    cerr = hipMemcpy(d_h,&value,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    ierr = PetscLogCpuToGpu(sizeof(PetscScalar));CHKERRQ(ierr);
    cerr = WaitForGPU();CHKERRCUDA(cerr);
    ierr = VecCUDARestoreArray(bv->buffer,&a);CHKERRQ(ierr);
  } else { /* cpu memory */
    h[bv->nc+j] = value;
  }
  PetscFunctionReturn(0);
}

/*
   BV_SquareSum_CUDA - Returns the value h'*h, where h represents the contents of the
   coefficients array (up to position j)
*/
PetscErrorCode BV_SquareSum_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscReal *sum)
{
  PetscErrorCode    ierr;
  const PetscScalar *d_h;
  PetscScalar       dot;
  PetscInt          i;
  PetscBLASInt      idx,one=1;
  hipblasStatus_t    cberr;
  hipblasHandle_t    cublasv2handle;

  PetscFunctionBegin;
  if (!h) {
    ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayRead(bv->buffer,&d_h);CHKERRQ(ierr);
    ierr = PetscBLASIntCast(bv->nc+j,&idx);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cberr = cublasXdotc(cublasv2handle,idx,d_h,one,d_h,one,&dot);CHKERRCUBLAS(cberr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(2.0*bv->nc+j);CHKERRQ(ierr);
    *sum = PetscRealPart(dot);
    ierr = VecCUDARestoreArrayRead(bv->buffer,&d_h);CHKERRQ(ierr);
  } else { /* cpu memory */
    *sum = 0.0;
    for (i=0;i<bv->nc+j;i++) *sum += PetscRealPart(h[i]*PetscConj(h[i]));
    ierr = PetscLogFlops(2.0*bv->nc+j);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#define X_AXIS        0
#define BLOCK_SIZE_X 64
#define TILE_SIZE_X  16 /* work to be done by any thread on axis x */

/*
   Set the kernels grid dimensions
   xcount: number of kernel calls needed for the requested size
 */
PetscErrorCode SetGrid1D(PetscInt n, dim3 *dimGrid, dim3 *dimBlock,PetscInt *xcount)
{
  PetscInt              one=1;
  PetscBLASInt          card;
  struct hipDeviceProp_t devprop;
  hipError_t           cerr;

  PetscFunctionBegin;
  *xcount = 1;
  if (n>BLOCK_SIZE_X) {
    dimBlock->x = BLOCK_SIZE_X;
    dimGrid->x = (n+BLOCK_SIZE_X*TILE_SIZE_X-one)/BLOCK_SIZE_X*TILE_SIZE_X;
  } else {
    dimBlock->x = (n+TILE_SIZE_X-one)/TILE_SIZE_X;
    dimGrid->x = one;
  }
  cerr = hipGetDevice(&card);CHKERRCUDA(cerr);
  cerr = hipGetDeviceProperties(&devprop,card);CHKERRCUDA(cerr);
  if (dimGrid->x>(unsigned)devprop.maxGridSize[X_AXIS]) {
    *xcount = (dimGrid->x+devprop.maxGridSize[X_AXIS]-one)/devprop.maxGridSize[X_AXIS];
    dimGrid->x = devprop.maxGridSize[X_AXIS];
  }
  PetscFunctionReturn(0);
}

/* pointwise multiplication */
__global__ void PointwiseMult_kernel(PetscInt xcount,PetscScalar *a,const PetscScalar *b,PetscInt n)
{
  PetscInt i,x;

  x = xcount*gridDim.x*blockDim.x+blockIdx.x*blockDim.x*TILE_SIZE_X+threadIdx.x*TILE_SIZE_X;
  for (i=x;i<x+TILE_SIZE_X&&i<n;i++) {
    a[i] *= PetscRealPart(b[i]);
  }
}

/* pointwise division */
__global__ void PointwiseDiv_kernel(PetscInt xcount,PetscScalar *a,const PetscScalar *b,PetscInt n)
{
  PetscInt i,x;

  x = xcount*gridDim.x*blockDim.x+blockIdx.x*blockDim.x*TILE_SIZE_X+threadIdx.x*TILE_SIZE_X;
  for (i=x;i<x+TILE_SIZE_X&&i<n;i++) {
    a[i] /= PetscRealPart(b[i]);
  }
}

/*
   BV_ApplySignature_CUDA - Computes the pointwise product h*omega, where h represents
   the contents of the coefficients array (up to position j) and omega is the signature;
   if inverse=TRUE then the operation is h/omega
*/
PetscErrorCode BV_ApplySignature_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscBool inverse)
{
  PetscErrorCode    ierr;
  PetscScalar       *d_h;
  const PetscScalar *d_omega,*omega;
  PetscInt          i,xcount;
  dim3              blocks3d, threads3d;
  hipError_t       cerr;

  PetscFunctionBegin;
  if (!(bv->nc+j)) PetscFunctionReturn(0);
  if (!h) {
    ierr = VecCUDAGetArray(bv->buffer,&d_h);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayRead(bv->omega,&d_omega);CHKERRQ(ierr);
    ierr = SetGrid1D(bv->nc+j,&blocks3d,&threads3d,&xcount);CHKERRQ(ierr);
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    if (inverse) {
      for (i=0;i<xcount;i++) {
        PointwiseDiv_kernel<<<blocks3d,threads3d>>>(i,d_h,d_omega,bv->nc+j);
      }
    } else {
      for (i=0;i<xcount;i++) {
        PointwiseMult_kernel<<<blocks3d,threads3d>>>(i,d_h,d_omega,bv->nc+j);
      }
    }
    cerr = hipGetLastError();CHKERRCUDA(cerr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    ierr = PetscLogGpuFlops(1.0*bv->nc+j);CHKERRQ(ierr);
    cerr = WaitForGPU();CHKERRCUDA(cerr);
    ierr = VecCUDARestoreArrayRead(bv->omega,&d_omega);CHKERRQ(ierr);
    ierr = VecCUDARestoreArray(bv->buffer,&d_h);CHKERRQ(ierr);
  } else {
    ierr = VecGetArrayRead(bv->omega,&omega);CHKERRQ(ierr);
    if (inverse) for (i=0;i<bv->nc+j;i++) h[i] /= PetscRealPart(omega[i]);
    else for (i=0;i<bv->nc+j;i++) h[i] *= PetscRealPart(omega[i]);
    ierr = VecRestoreArrayRead(bv->omega,&omega);CHKERRQ(ierr);
    ierr = PetscLogFlops(1.0*bv->nc+j);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*
   BV_SquareRoot_CUDA - Returns the square root of position j (counted after the constraints)
   of the coefficients array
*/
PetscErrorCode BV_SquareRoot_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscReal *beta)
{
  PetscErrorCode    ierr;
  const PetscScalar *d_h;
  PetscScalar       hh;
  hipError_t       cerr;

  PetscFunctionBegin;
  if (!h) {
    ierr = VecCUDAGetArrayRead(bv->buffer,&d_h);CHKERRQ(ierr);
    cerr = hipMemcpy(&hh,d_h+bv->nc+j,sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    ierr = PetscLogGpuToCpu(sizeof(PetscScalar));CHKERRQ(ierr);
    cerr = WaitForGPU();CHKERRCUDA(cerr);
    ierr = BV_SafeSqrt(bv,hh,beta);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayRead(bv->buffer,&d_h);CHKERRQ(ierr);
  } else {
    ierr = BV_SafeSqrt(bv,h[bv->nc+j],beta);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*
   BV_StoreCoefficients_CUDA - Copy the contents of the coefficients array to an array dest
   provided by the caller (only values from l to j are copied)
*/
PetscErrorCode BV_StoreCoefficients_CUDA(BV bv,PetscInt j,PetscScalar *h,PetscScalar *dest)
{
  PetscErrorCode    ierr;
  const PetscScalar *d_h,*d_a;
  PetscInt          i;
  hipError_t       cerr;

  PetscFunctionBegin;
  if (!h) {
    ierr = VecCUDAGetArrayRead(bv->buffer,&d_a);CHKERRQ(ierr);
    d_h = d_a + j*(bv->nc+bv->m)+bv->nc;
    cerr = hipMemcpy(dest-bv->l,d_h,(j-bv->l)*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    ierr = PetscLogGpuToCpu((j-bv->l)*sizeof(PetscScalar));CHKERRQ(ierr);
    cerr = WaitForGPU();CHKERRCUDA(cerr);
    ierr = VecCUDARestoreArrayRead(bv->buffer,&d_a);CHKERRQ(ierr);
  } else {
    for (i=bv->l;i<j;i++) dest[i-bv->l] = h[bv->nc+i];
  }
  PetscFunctionReturn(0);
}

