#include "hip/hip_runtime.h"
/*
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
   SLEPc - Scalable Library for Eigenvalue Problem Computations
   Copyright (c) 2002-2020, Universitat Politecnica de Valencia, Spain

   This file is part of SLEPc.
   SLEPc is distributed under a 2-clause BSD license (see LICENSE).
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
*/
/*
   SLEPc singular value solver: "cyclic" (CUDA implementation)
*/
#include <slepc/private/svdimpl.h>
#include "../src/svd/impls/cyclic/cyclic.h"

PetscErrorCode MatMult_Cyclic_CUDA(Mat B,Vec x,Vec y)
{
  PetscErrorCode ierr;
  SVD            svd;
  SVD_CYCLIC     *cyclic;
  PetscScalar    *d_px,*d_py;
  PetscInt       m;

  PetscFunctionBegin;
  ierr = MatShellGetContext(B,(void**)&svd);CHKERRQ(ierr);
  cyclic = (SVD_CYCLIC*)svd->data;
  ierr = SVDMatGetLocalSize(svd,&m,NULL);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(x,(const PetscScalar**)&d_px);CHKERRQ(ierr);
  ierr = VecCUDAGetArray(y,&d_py);CHKERRQ(ierr);
  ierr = VecCUDAPlaceArray(cyclic->x1,d_px);CHKERRQ(ierr);
  ierr = VecCUDAPlaceArray(cyclic->x2,d_px+m);CHKERRQ(ierr);
  ierr = VecCUDAPlaceArray(cyclic->y1,d_py);CHKERRQ(ierr);
  ierr = VecCUDAPlaceArray(cyclic->y2,d_py+m);CHKERRQ(ierr);
  ierr = SVDMatMult(svd,PETSC_FALSE,cyclic->x2,cyclic->y1);CHKERRQ(ierr);
  ierr = SVDMatMult(svd,PETSC_TRUE,cyclic->x1,cyclic->y2);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(cyclic->x1);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(cyclic->x2);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(cyclic->y1);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(cyclic->y2);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(x,(const PetscScalar**)&d_px);CHKERRQ(ierr);
  ierr = VecCUDARestoreArray(y,&d_py);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatCreateVecs_Cyclic_CUDA(Mat B,Vec *right,Vec *left)
{
  PetscErrorCode ierr;
  SVD            svd;
  SVD_CYCLIC     *cyclic;
  PetscInt       M,N,m,n;
  PetscMPIInt    size;

  PetscFunctionBegin;
  ierr = MatShellGetContext(B,(void**)&svd);CHKERRQ(ierr);
  cyclic = (SVD_CYCLIC*)svd->data;
  ierr = SVDMatGetSize(svd,&M,&N);CHKERRQ(ierr);
  ierr = SVDMatGetLocalSize(svd,&m,&n);CHKERRQ(ierr);
  ierr = MPI_Comm_size(PetscObjectComm((PetscObject)cyclic->mat),&size);CHKERRQ(ierr);
  if (right) {
    ierr = VecCreate(PetscObjectComm((PetscObject)cyclic->mat),right);CHKERRQ(ierr);
    ierr = VecSetSizes(*right,m+n,M+N);CHKERRQ(ierr);
    if (size>1) {
      ierr = VecSetType(*right,VECMPICUDA);CHKERRQ(ierr);
    } else {
      ierr = VecSetType(*right,VECSEQCUDA);CHKERRQ(ierr);
    }
  }
  if (left) {
    ierr = VecCreate(PetscObjectComm((PetscObject)cyclic->mat),left);CHKERRQ(ierr);
    ierr = VecSetSizes(*left,m+n,M+N);CHKERRQ(ierr);
    if (size>1) {
      ierr = VecSetType(*left,VECMPICUDA);CHKERRQ(ierr);
    } else {
      ierr = VecSetType(*left,VECSEQCUDA);CHKERRQ(ierr);
    }
  }
  PetscFunctionReturn(0);
}

